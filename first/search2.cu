#include "hip/hip_runtime.h"
/*

*/

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <math.h>
#include <float.h>
//#include <hip/hip_runtime.h>
#include "unimem.h"
#include "first.h"

//#define DEBUG_CUDA
#define MAX_RESULTS 100000
//#define PACK_RESULTS

struct boundaries {
	coord x_min;
	coord y_min;
	coord x_max;
	coord y_max;
	bool intersection;
};

struct node* m_dev_node = NULL;
__constant__ boundaries dev_bonds[1];
__constant__ unsigned dev_threads_count[1];
unsigned m_threads_count;
indexer m_count_branches;
int m_length_of_tree = 0;
int m_multi_processor_count = 1;
int m_warp_size2 = 64;

//__constant__ struct branch *m_ttt_cuda_first_branch = NULL;

__device__ bool cuda_check_intersection(coord p1x, coord p1y, coord p2x, coord p2y, coord p3x, coord p3y, coord p4x, coord p4y);
__device__ coord cuda_distance(coord px, coord py, coord line_p0x, coord line_p0y, coord line_p1x, coord line_p1y);

#ifdef PACK_RESULTS
/// compare
int cmp(const void* a, const void* b)
{
	return (int)(*(indexer*)a - *(indexer*)b);
}

#endif

extern "C"
bool init_cuda_device(int deviceID, struct node* node)
{
	if (!node)
		return false;
	//return false;

	struct node *nd = node;
	unsigned count1[64], i = 0;
	m_count_branches = 0;
	for (int j = 0; j < 64; j++) count1[j] = 0;
	//count1[0] = 1;

	struct node *stack_node[64];
	int stack_pos = 0;
	indexer stack_idx[64];
	alignas(16) struct branch *first_branch = NULL;
	alignas(16) struct node* stack_first_node[64];
	for (unsigned i = 0; i < 64; ++i) {
		stack_first_node[i] = NULL;
	}
	while (i < nd->count_child_nodes) {
		if (!stack_first_node[stack_pos] || nd < stack_first_node[stack_pos])
			stack_first_node[stack_pos] = nd;
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				if (!first_branch || br < first_branch)
					first_branch = br;
			}
			/*if (!count_br)
				count_br = nd->count_child_nodes;
			if (count_br != nd->count_child_nodes)
				printf("Branches %u vs %u\n", count_br, nd->count_child_nodes);*/
			m_count_branches += nd->count_child_nodes;
				// return from stack
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
				else {
					//if (count1[stack_pos]) {
						//if (count1[stack_pos] != nd->count_child_nodes) {
						//	printf("Nodes %u vs %u\n", count1[stack_pos], nd->count_child_nodes);
						//}
						count1[stack_pos] += nd->count_child_nodes;
					//}
					//else {
					//	count1[stack_pos] = nd->count_child_nodes;
					//}
				}
			}
		}
		else {
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			i = 0;
			nd = (struct node*)nd->child_node[i];
			/*if (!count1[stack_pos])
				count1[stack_pos] = nd->count_child_nodes;
			else
				count1[stack_pos] += nd->count_child_nodes;*/
		}
		/*} else if (i < nd->count_child_nodes) {
		i++;*/
		/*if (!count1[stack_pos])
			count1[stack_pos] = nd->count_child_nodes;
		else
			count1[stack_pos] += nd->count_child_nodes;
			*/
	}

	//return false;

	int deviceCount;
	hipError_t er1 = hipGetDeviceCount(&deviceCount);
	printf("DevicecheckCudaErrors Count: %d\n", deviceCount);

	if (deviceID == -1)
		deviceID = 0;

	hipDeviceProp_t prop;
	for (int ii = 0; ii < deviceCount; ++ii) {
		er1 = hipGetDeviceProperties(&prop, ii);
		if (prop.major < 2 || prop.canMapHostMemory != 1)
		{
			printf("ERROR: calculation requires GPU devices with compute SM 2.0 or higher, or can not using MapHostMemory.\n");
			printf("Current GPU device has compute SM%d.%d, Exiting...", prop.major, prop.minor);
			//exit(EXIT_WAIVED);
			return false;
		}

		printf("GPU device name is %s\n", prop.name);
		printf("GPU total memory = %.0f Mb\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		printf("Number of multiprocessors on the device = %u\n", prop.multiProcessorCount);
	}

	er1 = hipSetDevice(deviceID);
	hipSetDeviceFlags(hipDeviceMapHost);
	er1 = hipGetDeviceProperties(&prop, deviceID);
	m_multi_processor_count = prop.multiProcessorCount;
	m_warp_size2 = prop.warpSize * 2;
	m_threads_count = prop.multiProcessorCount * prop.warpSize * 2;
	//er1 = hipMalloc((void**)&dev_threads_count, sizeof(unsigned));
	er1 = hipMemcpyToSymbol(HIP_SYMBOL(dev_threads_count), &m_threads_count, sizeof(unsigned));

	// copy rtree
	int pos = 63;

	for (; pos >= 0; --pos) {
		if (count1[pos])
			break;
	}
	m_length_of_tree = pos + 1;

	// allocationg memory for branches
	alignas(16) struct branch* tbr = (struct branch*)aligned_alloc(16, sizeof(struct branch) * m_count_branches);
	// struct branch* first_branch = NULL;
	/*nd = node;
	i = 0;
	unsigned k = 0;
	while (i < nd->count_child_nodes) {
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				//if (!first_branch || br < first_branch)
				//	first_branch = br;
				memcpy(tbr + k, br, sizeof(struct branch));
				k++;
			}
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					// insert to stack
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
			}
		}
		else {
			// insert to stack
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			nd = (struct node*)nd->child_node[i];
		}
	}*/
	/*nd = node;
	i = 0;
	for (int j = 0; j <= pos; ++j)
		nd = (struct node*)(nd->child_node[0]);
	for (indexer j = 0; j < count1[pos]; ++j) {
		for (indexer k = 0; k < nd[j].count_child_nodes; ++k) {
			memcpy(tbr + i, nd[j].child_node[k], sizeof(struct branch));
			i++;
		}
	}
	*/
	memcpy(tbr, first_branch, sizeof(struct branch) * m_count_branches);

	// for debug
	/*printf("\n\n\n======================================================================\n");
	for (indexer i = 0; i < count1[pos]; ++i) {
		if ((struct node*)(stack_first_node[pos + 1])[i].is_last_node) {
			unsigned tt = ((struct node*)(stack_first_node[pos + 1]))[i].count_child_nodes;
			for (indexer ii = 0; ii < tt; ++ii) {
				unsigned idx = (struct branch*)((struct node*)(stack_first_node[pos + 1])[i].child_node[ii]) - first_branch;
				if (!idx)
					printf("0\n");
				else
					printf("%u\n", idx);
			}
		}
		else {
			printf("Error last node %u\n", i);
		}
	}*/

	// copy data of branches to device
	clock_t t1 = clock();
	hipStream_t stream;
	hipStreamCreate(&stream);
	for (indexer i = 0; i < m_count_branches; ++i)
	{
		void *data_ptr = tbr[i].leaf_x;
		er1 = hipMalloc((void**)&(tbr[i].leaf_x), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_x, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_y;
		er1 = hipMalloc((void**)&(tbr[i].leaf_y), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_y, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_number;
		er1 = hipMalloc((void**)&(tbr[i].leaf_number), sizeof(indexer) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_number, data_ptr, sizeof(indexer) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].merge_next_leaf;
		er1 = hipMalloc((void**)&(tbr[i].merge_next_leaf), sizeof(bool) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].merge_next_leaf, data_ptr, sizeof(bool) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		/*data_ptr = tbr[i].xsh_min;
		er1 = hipMalloc((void**)&(tbr[i].xsh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].xsh_max;
		er1 = hipMalloc((void**)&(tbr[i].xsh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_min;
		er1 = hipMalloc((void**)&(tbr[i].ysh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_max;
		er1 = hipMalloc((void**)&(tbr[i].ysh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream); */
		data_ptr = tbr[i].offset;
		er1 = hipMalloc((void**)&(tbr[i].offset), sizeof(indexer) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].offset, data_ptr, sizeof(indexer) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
	}
	er1 = hipStreamSynchronize(stream);
	er1 = hipStreamDestroy(stream);
	clock_t t2 = clock();
	printf("Time copying data to device = %u ms\n", t2 - t1);

	// copy branches to device
	struct branch *dev_br = NULL;
	er1 = hipMalloc((void**)&dev_br, sizeof(struct branch) * m_count_branches);
	er1 = hipMemcpy(dev_br, tbr, sizeof(struct branch) * m_count_branches, hipMemcpyHostToDevice);
	//hipMemcpyToSymbol(HIP_SYMBOL(m_ttt_cuda_first_branch), &dev_br, sizeof(struct branch*));

	//return false;
	alignas(16) struct node *to_dev_nd[65];
	//void **to_dev_child[64];
	struct node *dev_nd = NULL, *dev_nd_prev = NULL, *dev_ptr = NULL;
	// to_dev_nd[0] = (struct node*)aligned_alloc(16, sizeof(struct node));
	// memcpy(to_dev_nd[0], nd, sizeof(struct node));
	struct node* tnd = node;
	//for (unsigned j = 0; j <= pos; ++j)
	//	tnd = (struct node*)(tnd->child_node[0]);
	//unsigned j = 0;
	//void* tmp1 = NULL;
	unsigned count = tnd->count_child_nodes, prev_count = 1;
	//printf("\n\n\n======================================================================\n");
	for (int k1 = pos; k1 >= 0; --k1) {
		tnd = node;
		//for (unsigned j = 0; j <= k1; ++j)
		//	tnd = (struct node*)(tnd->child_node[0]);
		// data child node
		to_dev_nd[k1] = (struct node*)aligned_alloc(16, sizeof(struct node) * count1[k1]);
		//memcpy(to_dev_nd[k1], tnd/*->child_node[0]*/, sizeof(struct node) * count1[k1]);
		memcpy(to_dev_nd[k1], stack_first_node[k1 + 1], sizeof(struct node) * count1[k1]);
		// pointer to child_node on host
		for (indexer j = 0; j < count1[k1]; ++j) {
			//(to_dev_nd[k1])[j]->child_node = (void**)aligned_alloc(16, sizeof(void*) * MAX_NODES); // tnd->count_child_nodes);
			//(to_dev_child[k1])[j] = to_dev_nd[j]->child_node;
			dev_ptr = NULL;
			er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * MAX_NODES);
			(to_dev_nd[k1])[j].child_node = (void**)dev_ptr;
			for (indexer k2 = 0; k2 < MAX_NODES; ++k2) {
				if (k1 == pos) {
					// copy pointer of branches
					//struct branch *ptr = &(dev_br[k2 + j * MAX_NODES]);
					unsigned idx = (struct branch*)((struct node*)(stack_first_node[k1 + 1])[j].child_node[k2]) - first_branch;
					//if (idx == 4899)
					//	printf("%u\n", idx);
					struct branch *ptr = &(dev_br[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct branch*), hipMemcpyHostToDevice);
				}
				else {
					// copy pointer of nodes
					//struct node* ptr = &(dev_nd_prev[k2 + j * MAX_NODES]);
					unsigned idx = (struct node*)(stack_first_node[k1 + 1])[j].child_node[k2] - (struct node*)(stack_first_node[k1 + 2]);
					//printf("%u\n", idx);
					struct node *ptr = &(dev_nd_prev[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
				}
			}
		}
		//printf("==========================================\n\n\n");
		// pointers of child nodes
		er1 = hipMalloc((void**)&dev_nd, sizeof(struct node) * count1[k1]); // tnd->count_child_nodes);
		hipMemcpy(dev_nd, to_dev_nd[k1], sizeof(struct node) * count1[k1], hipMemcpyHostToDevice);
		dev_nd_prev = dev_nd;
	}
	// copy top node (root)
	to_dev_nd[64] = (struct node*)aligned_alloc(16, sizeof(struct node));
	memcpy(to_dev_nd[64], node/*->child_node[0]*/, sizeof(struct node));
	dev_ptr = NULL;
	er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * node->count_child_nodes);
	(to_dev_nd[64])[0].child_node = (void**)dev_ptr;
	for (indexer k2 = 0; k2 < node->count_child_nodes; ++k2) {
		// copy pointer of nodes
		//struct node* ptr = &(dev_nd_prev[k2]);
		unsigned idx = (struct node*)(stack_first_node[0])[0].child_node[k2] - (struct node*)(stack_first_node[1]);
		struct node* ptr = &(dev_nd_prev[idx]);
		er1 = hipMemcpy((void*)((to_dev_nd[64])[0].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
	}
	// pointers of child nodes
	er1 = hipMalloc((void**)&dev_nd, sizeof(struct node)); // tnd->count_child_nodes);
	er1 = hipMemcpy(dev_nd, to_dev_nd[64], sizeof(struct node), hipMemcpyHostToDevice);
	m_dev_node = dev_nd;
	printf("============== 0x%llx, 0x%llx, prev = 0x%llx\n", m_dev_node, dev_nd, dev_nd_prev);

	// free memory
	for (int k1 = pos; k1 >= 0; --k1) {
		_aligned_free(to_dev_nd[k1]);
	}
	_aligned_free(to_dev_nd[64]);

	// allocating memory for root
	//er1 = hipMalloc((void**)&m_dev_node, sizeof(struct node));
	// copy to device root of tree
	//hipMemcpy(m_dev_node, to_dev_nd[0], sizeof(struct node), hipMemcpyHostToDevice);

	return true;
}

extern "C"
bool destroy_cuda_device()
{
	//hipFree(dev_threads_count);
	hipError_t er1 = hipDeviceReset();
	return er1 == hipSuccess ? true : false;
}

#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
extern "C"
indexer* cuda_search_rect2(/*in*/struct node *nd, /*in*/coord x_min, /*in*/coord y_min, /*in*/coord x_max, /*in*/coord y_max, bool /*in*/intersection, /*out*/indexer *count_items);
/* searchin items in selected rectangle on cuda device imlementation (nodes) */
__global__ void cuda_search_rect2_impl1(void **nd, indexer *iter_count, indexer *atomic_iter, /*out*/ void **next_nd);
/* searchin items in selected rectangle on cuda device imlementation (branches) */
__global__ void cuda_search_rect2_impl2(void **br_ptr, indexer *atomic_iter, /*out*/ indexer *idxs);

/* searching the nearest item to point in selected radius */
extern "C"
indexer* cuda_search_nearest_item2(/*in*//*struct node *nd,*/ /*in*/coord x, /*in*/coord y, /*in*/coord radius, bool intersection, /*out*/coord *dist);
/* searching the nearest item on device implementation (step 2) */
__global__ void cuda_search_nearest_item2_impl2(void **br_ptr, /*indexer *atomic_iter,*/ coord x, coord y, /*out*/ indexer *idxs, /*out*/ coord *dist);
/* searching the nearest item on device implementation (step 3) */
__global__ void cuda_search_nearest_item2_impl3(/*in*/ indexer *idxs, /*in*/ coord *dist, /*in*/indexer count, /*in*/indexer *atomic_iter, /*out*/ indexer *idxs2, /*out*/ coord *dist2);

#else
extern "C"
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
__global__ indexer* search_rect2_impl(void *nd_ptr, indexer iter_count, /*out*/indexer *count_items)
#endif // CALC_POINT

#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
indexer* cuda_search_rect2(node * nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, indexer * count_items)
#else
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
#endif // CALC_POINT
{

	// memory for result
	/*size_t mem_size = 256;
	size_t count_mem = 1;
	alignas(16) indexer* idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * mem_size * count_mem); */
	hipError_t er1;
	hipStream_t stream;
	hipStreamCreate(&stream);

	/*indexer *host_idxs = NULL, *dev_idxs = NULL; // , *dev_tmp_idxs = NULL;;
	hipHostAlloc((void**)&host_idxs, sizeof(indexer) * MAX_RESULTS, hipHostMallocMapped);
	hipHostGetDevicePointer((void**)&dev_idxs, host_idxs, 0);
	//hipMalloc((void**)&dev_tmp_idxs, sizeof(indexer) * MAX_RESULTS);
	*/
	indexer *dev_idxs = NULL;
	hipMalloc((void**)&dev_idxs, sizeof(indexer) * MAX_RESULTS);

	// searching
	hipEvent_t start, stop;
	float gtime = 0.0;
	int device_id;
	/*hipDeviceProp_t prop;
	er1 = hipGetDevice(&device_id);
	er1 = hipGetDeviceProperties(&prop, device_id);
	dim3 grid_size = dim3(prop.multiProcessorCount, 1, 1), block_size = dim3(prop.warpSize * 2, 1, 1);
	*/
	dim3 grid_size = dim3(m_multi_processor_count, 1, 1), block_size = dim3(m_warp_size2, 1, 1);
	// store boundaries
	boundaries b1;
	b1.intersection = intersection; b1.x_max = x_max; b1.x_min = x_min; b1.y_max = y_max; b1.y_min = y_min;
	//hipMalloc((void**)dev_bonds, sizeof(struct boundaries));
	er1 = hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_bonds), &b1, sizeof(struct boundaries), 0, hipMemcpyHostToDevice, stream);
	// for store count of iterations to next step
	indexer *dev_atomic_iter = NULL;
	er1 = hipMalloc((void**)&dev_atomic_iter, sizeof(indexer));
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	// store pointers for next step
	void **dev_ptr = NULL, **dev_ptr2 = NULL;
	er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx, count_br = %u\n", &m_dev_node, m_dev_node, m_count_branches);
	void **tptr = (void**)(&m_dev_node);
	er1 = hipMemcpyAsync(dev_ptr, tptr, sizeof(void*), hipMemcpyHostToDevice, stream);
	er1 = hipMalloc((void**)&dev_ptr2, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx; dev_ptr = 0x%llx\n", &m_dev_node, m_dev_node, dev_ptr);
	// count items
	//indexer *dev_count_items = NULL;
	//er1 = hipMalloc((void**)&dev_count_items, sizeof(indexer));
	// count of iterations
	indexer atomic_iter = 1;
	indexer *dev_iter_count = NULL;
	er1 = hipMalloc((void**)&dev_iter_count, sizeof(indexer));
	//er1 = hipMemsetAsync(dev_iter_count, 0, sizeof(indexer), stream);
	//er1 = hipMemsetAsync(dev_iter_count, 1, 1, stream);
	er1 = hipMemcpyAsync(dev_iter_count, &atomic_iter, sizeof(indexer), hipMemcpyHostToDevice);
	er1 = hipStreamSynchronize(stream);

#ifdef DEBUG_CUDA
	clock_t t1 = clock();
	er1 = hipEventCreate(&start);
	er1 = hipEventCreate(&stop);
	er1 = hipEventRecord(start, stream);
#endif
	
	// calculating nodes
	for (int i = 0; i < m_length_of_tree + 1; ++i) {
		er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
		if (atomic_iter > m_warp_size2 /*prop.warpSize * 2 */) {
			unsigned t = (unsigned)ceil((double)atomic_iter / (double)(m_warp_size2 /*prop.warpSize * 2.0 */));
			block_size = dim3(m_warp_size2 /*prop.warpSize * 2 */, 1, 1);
			grid_size = dim3(t, 1, 1);
		}
		else {
			grid_size = dim3(1, 1, 1);
			block_size = dim3(atomic_iter, 1, 1);
		}
		cuda_search_rect2_impl1 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_iter_count, dev_atomic_iter, dev_ptr2);

		er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
		er1 = hipMemcpyAsync(dev_ptr, dev_ptr2, sizeof(void*) * atomic_iter, hipMemcpyDeviceToDevice, stream);
		er1 = hipMemcpyAsync(dev_iter_count, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToDevice, stream);
		hipStreamSynchronize(stream);
		//printf("===== Iter %i: next = %u (%s)\n", i, atomic_iter, er1 == hipSuccess ? "true" : "false");
		//hipDeviceSynchronize();
	}
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(stop, stream);
	er1 = hipEventSynchronize(stop);
	er1 = hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 1 time = %f ms\n", gtime);
#endif

	// calculating branches
	grid_size = dim3(atomic_iter, 1, 1);
	block_size = dim3(m_warp_size2 /*prop.warpSize * 2 */, 1, 1);
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(start, stream);
#endif
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	cuda_search_rect2_impl2 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_atomic_iter, dev_idxs);
	er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
	er1 = hipStreamSynchronize(stream);
	//er1 = hipDeviceSynchronize();
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(stop, stream);
	er1 = hipEventSynchronize(stop);
	er1 = hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 2 time = %f ms\n", gtime);
	clock_t t2 = clock();
	printf("All kernels time = %i ms\n", t2 - t1);
#endif

	er1 = hipMemcpyAsync(count_items, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
	indexer *idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * *count_items);
	//er1 = hipMemcpyAsync(idxs, host_idxs, sizeof(indexer) * *count_items, hipMemcpyHostToHost, stream);
	er1 = hipMemcpyAsync(idxs, dev_idxs, sizeof(indexer) * *count_items, hipMemcpyDeviceToHost, stream);
	er1 = hipStreamSynchronize(stream);
#ifdef DEBUG_CUDA
	printf("Total results from device = %u\n", *count_items);
#endif

	// freeing and destroying
	hipStreamDestroy(stream);

	er1 = hipFree(dev_iter_count);
	er1 = hipFree(dev_ptr);
	er1 = hipFree(dev_ptr2);
	//er1 = hipFree(dev_tmp_idxs);
	//er1 = hipFree(dev_count_items);
	er1 = hipFree(dev_atomic_iter);
	//er1 = hipHostFree(host_idxs);
	er1 = hipFree(dev_idxs);
#ifdef DEBUG_CUDA
	er1 = hipEventDestroy(stop);
	er1 = hipEventDestroy(start);
#endif

#ifdef PACK_RESULTS
	if (*count_items) {
		qsort(idxs, *count_items, sizeof(indexer), cmp);
		indexer j = 1;
		indexer offset = 0;
		for (indexer i = 0; i < *count_items - 1 - offset; ++i) {
			//if (idxs[i] == 3617359)
			//	idxs[i] = 3617359;
			if (idxs[i] == idxs[i + 1 + offset]) {
				offset++;
				idxs[i + 1] = idxs[i + 1 + offset];
				i--;
				continue;
			}
			if (offset)
				idxs[i + 1] = idxs[i + 1 + offset];
			j++;
		}
		*count_items = j;
		idxs = (indexer*)_aligned_realloc(idxs, sizeof(indexer) * j, 16);
	}
#endif
	return idxs;
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl1(void **nd_ptr, indexer *iter_count, indexer *atomic_iter, /*out*/ void** next_nd)
{
	int idxx = threadIdx.x;

	// to temporary store node index
	/*__shared__ indexer store[64];
	store[threadIdx.x] = (indexer)-1;
	__shared__ int store_idx[1];
	if (!threadIdx.x)
		store_idx[threadIdx.x] = 0;
		*/

	struct node** nd = (struct node**)nd_ptr;
	//indexer idx = 0;

#ifdef CALC_POINT
	//coord tmp_dist = FLT_MAX;
	//indexer tmp_idx = -1;
#endif // CALC_POINT
	indexer curr_indexer = idxx + blockIdx.x * blockDim.x; // (*dev_threads_count);
	if (curr_indexer < *iter_count) {
		struct node *curr_nd = nd[curr_indexer];
		__shared__ coord nd_x1[64], nd_x2[64], nd_y1[64], nd_y2[64];
		nd_x1[threadIdx.x] = curr_nd->x1;
		nd_x2[threadIdx.x] = curr_nd->x2;
		nd_y1[threadIdx.x] = curr_nd->y1;
		nd_y2[threadIdx.x] = curr_nd->y2;
			
		// node in bounrary or bounrary in node
		if (nd_x1[threadIdx.x] <= dev_bonds->x_max && nd_x2[threadIdx.x] >= dev_bonds->x_min && nd_y1[threadIdx.x] <= dev_bonds->y_max && nd_y2[threadIdx.x] >= dev_bonds->y_min) {
				// node isn't fully in the boundary, than add to calculation to next iteration
				indexer t3 = atomicAdd(atomic_iter, curr_nd->count_child_nodes);
				//printf("Increase %i: %u to %u (%u)\n", idxx, t1, *atomic_iter, nd[idxx]->count_child_nodes);
				for (unsigned k = t3, t2 = 0; k < t3 + curr_nd->count_child_nodes; ++k, ++t2) {
					next_nd[k] = curr_nd->child_node[t2];
					//printf("Next index = %u\n", (struct branch*)(nd[curr_indexer]->child_node[t2]) - m_ttt_cuda_first_branch);
				}
		}
		else {
			// node and boundary isn't intersection
		}
	}
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl2(void **br_ptr, indexer *atomic_iter, /*out*/ indexer *idxs)
{
	int idxx = threadIdx.x;
	int idx_gr_br = blockIdx.x;

	// for store temporary results
	__shared__ indexer temp_res[65]; // must be as blockDim.x size + 1 (for rpevious result)
	__shared__ char temp_res_flag[64];
	temp_res[idxx] = (indexer)-1;
	temp_res_flag[idxx] = -1;
	if (!idxx)
		temp_res[64] = (indexer)-1;

	__shared__ coord leaf_x[65];
	__shared__ coord leaf_y[65];

	struct branch** br = (struct branch**)br_ptr;
	struct branch *curr_br = br[idx_gr_br];
	__syncthreads();

	__shared__ indexer start_num[1];
	if (!idxx)
		start_num[0] = curr_br->leaf_number[0];
		//start_num[0] = ((branch*)((struct branch**)br_ptr)[idx_gr_br])->leaf_number[0];
	__syncthreads();
	//if (start_num[0] != ((branch*)((struct branch**)br_ptr))->leaf_number[0]) {
	//	start_num[0] = ((branch*)((struct branch**)br_ptr)[idx_gr_br])->leaf_number[0];
	//}

	if (curr_br->x_min <= dev_bonds->x_max && curr_br->x_max >= dev_bonds->x_min && curr_br->y_min <= dev_bonds->y_max && curr_br->y_max >= dev_bonds->y_min) {
		int t = (int)ceilf((float)curr_br->count_leafs / (float)blockDim.x);
		for (int j = 0; j < t; ++j) {
			int curr_idx = idxx + j * blockDim.x; // curr_offset;
			if (/*j == t1 && */curr_idx < curr_br->count_leafs) {
				// loading frequantly using data
				leaf_x[idxx] = curr_br->leaf_x[curr_idx];
				leaf_y[idxx] = curr_br->leaf_y[curr_idx];
				if (!idxx && curr_idx + 64 < curr_br->count_leafs && curr_br->merge_next_leaf[curr_idx + 63]) {
					leaf_x[64] = curr_br->leaf_x[curr_idx + 64];
					leaf_y[64] = curr_br->leaf_y[curr_idx + 64];
				}
				//if (curr_br->leaf_number[curr_idx] == 3617359)
				//	curr_idx = curr_idx;

				// check points to enter in boundary
				if (leaf_x[idxx] >= dev_bonds->x_min && leaf_x[idxx] <= dev_bonds->x_max && leaf_y[idxx] >= dev_bonds->y_min && leaf_y[idxx] <= dev_bonds->y_max) {
					temp_res[idxx] = curr_br->leaf_number[curr_idx];
				}
				else if(dev_bonds->intersection) {
					bool fl1 = false;

					if (curr_br->merge_next_leaf[curr_idx]) {
						// last check: intersection

						// side 1/2
						fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x[idxx + 1], leaf_y[idxx + 1], dev_bonds->x_min, dev_bonds->y_min, dev_bonds->x_max, dev_bonds->y_min);

						// side 2/3
						if (!fl1) {
							fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x[idxx + 1], leaf_y[idxx + 1], dev_bonds->x_max, dev_bonds->y_min, dev_bonds->x_max, dev_bonds->y_max);
						}

						// side 3/4
						if (!fl1) {
							fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x[idxx + 1], leaf_y[idxx + 1], dev_bonds->x_max, dev_bonds->y_max, dev_bonds->x_min, dev_bonds->y_max);
						}

						// side 4/1
						if (!fl1) {
							fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x[idxx + 1], leaf_y[idxx + 1], dev_bonds->x_min, dev_bonds->y_max, dev_bonds->x_min, dev_bonds->y_min);
						}
					}
					else {
						indexer curr_num = curr_br->offset[curr_br->leaf_number[curr_idx] - start_num[0]];
						__shared__ coord leaf_x_offset[64];
						__shared__ coord leaf_y_offset[64];
						leaf_x_offset[idxx] = curr_br->leaf_x[curr_num];
						leaf_y_offset[idxx] = curr_br->leaf_y[curr_num];

						// side 1/2
						fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x_offset[idxx], leaf_y_offset[idxx], dev_bonds->x_min, dev_bonds->y_min, dev_bonds->x_max, dev_bonds->y_min);

						// side 2/3
						if (!fl1) {
							fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x_offset[idxx], leaf_y_offset[idxx], dev_bonds->x_max, dev_bonds->y_min, dev_bonds->x_max, dev_bonds->y_max);
						}

						// side 3/4
						if (!fl1) {
							fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x_offset[idxx], leaf_y_offset[idxx], dev_bonds->x_max, dev_bonds->y_max, dev_bonds->x_min, dev_bonds->y_max);
						}

						// side 4/1
						if (!fl1) {
							fl1 = cuda_check_intersection(leaf_x[idxx], leaf_y[idxx], leaf_x_offset[idxx], leaf_y_offset[idxx], dev_bonds->x_min, dev_bonds->y_max, dev_bonds->x_min, dev_bonds->y_min);
						}
					}

					if (fl1)
						temp_res[idxx] = curr_br->leaf_number[curr_idx];
				}
				__syncthreads();

				// packing temporary results
				if (temp_res[idxx] == temp_res[idxx + 1]) {
					__threadfence();
					temp_res[idxx + 1] = -1;
				}
				else {
					//__threadfence();
				}
				__syncthreads();

				if (!idxx) {
					if (temp_res[64] == temp_res[0]) {
						temp_res[0] = -1;
					}
				}
				//__syncthreads();

				// store temporary results to global array2
				if (temp_res[idxx] != -1) {
					int t2 = atomicAdd(atomic_iter, 1);
					if (t2 >= MAX_RESULTS - 1) {
						// can not store result
						atomicSub(atomic_iter, 1);
					}
					else {
						// can store result (idxs2 - temporary)
						idxs[t2] = temp_res[idxx];
						temp_res_flag[idxx] = idxx;
					}
				}

				__syncthreads();

				// store previous result
				for (int t2 = blockDim.x / 2; t2 > 0; t2 >>= 1)
				{
					if (idxx < t2) {
						if (temp_res_flag[idxx] < temp_res_flag[idxx + t2])
							temp_res_flag[idxx] = temp_res_flag[idxx + t2];
					}
					__syncthreads();
				}
				if (!idxx) {
					if (temp_res_flag[idxx] != -1) {
						temp_res[64] = temp_res[temp_res_flag[idxx]]; // idxs[idxx];
					}
				}

				// reset temporary resulats
				temp_res[idxx] = (indexer)-1;
				//temp_res2[idxx] = -1;
				temp_res_flag[idxx] = -1;
				__syncthreads();
			}
		}
	}
}

__device__ bool cuda_check_intersection(coord p1x, coord p1y, coord p2x, coord p2y, coord p3x, coord p3y, coord p4x, coord p4y)
{
	coord x4x3 = p4x - p3x;
	coord y4y3 = p4y - p3y;
	coord x1x3 = p1x - p3x;
	coord y1y3 = p1y - p3y;

	coord x2x3 = p2x - p3x;
	coord y2y3 = p2y - p3y;

	coord x2x1 = p2x - p1x;
	coord y2y1 = p2y - p1y;
	coord x3x1 = p3x - p1x;
	coord y3y1 = p3y - p1y;

	coord x4x1 = p4x - p1x;
	coord y4y1 = p4y - p1y;

	coord v1 = x4x3 * y1y3 - x1x3 * y4y3;
	coord v2 = x4x3 * y2y3 - x2x3 * y4y3;
	coord v3 = x2x1 * y3y1 - x3x1 * y2y1;
	coord v4 = x2x1 * y4y1 - x4x1 * y2y1;

	coord v1t = v1 * v2;
	coord v2t = v3 * v4;

	//if ((signbit(v1t) || v1t == 0.0) && (signbit(v2t) || v2t == 0.0)) {
	if (v1t <= 0.0 && v2t <= 0.0) {
		return true;
	}
	return false;
}

/* searching the nearest item to point in selected radius */
indexer* cuda_search_nearest_item2(/*in*//*struct node *nd,*/ /*in*/coord x, /*in*/coord y, /*in*/coord radius, bool intersection, /*out*/coord *dist)
{
	hipError_t er1;
	hipStream_t stream;
	hipStreamCreate(&stream);

	// searching
	hipEvent_t start, stop;
	float gtime = 0.0;

	dim3 grid_size = dim3(m_multi_processor_count, 1, 1), block_size = dim3(m_warp_size2, 1, 1);
	// store boundaries
	boundaries b1;
	b1.intersection = intersection; b1.x_max = x + radius; b1.x_min = x - radius; b1.y_max = y + radius; b1.y_min = y - radius;
	//hipMalloc((void**)dev_bonds, sizeof(struct boundaries));
	er1 = hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_bonds), &b1, sizeof(struct boundaries), 0, hipMemcpyHostToDevice, stream);
	// for store count of iterations to next step
	indexer *dev_atomic_iter = NULL;
	er1 = hipMalloc((void**)&dev_atomic_iter, sizeof(indexer));
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	// store pointers for next step
	void **dev_ptr = NULL, **dev_ptr2 = NULL;
	er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx, count_br = %u\n", &m_dev_node, m_dev_node, m_count_branches);
	void **tptr = (void**)(&m_dev_node);
	er1 = hipMemcpyAsync(dev_ptr, tptr, sizeof(void*), hipMemcpyHostToDevice, stream);
	er1 = hipMalloc((void**)&dev_ptr2, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx; dev_ptr = 0x%llx\n", &m_dev_node, m_dev_node, dev_ptr);
	// count items
	//indexer *dev_count_items = NULL;
	//er1 = hipMalloc((void**)&dev_count_items, sizeof(indexer));
	// count of iterations
	indexer atomic_iter = 1;
	indexer *dev_iter_count = NULL;
	er1 = hipMalloc((void**)&dev_iter_count, sizeof(indexer));
	er1 = hipMemcpyAsync(dev_iter_count, &atomic_iter, sizeof(indexer), hipMemcpyHostToDevice);
	er1 = hipStreamSynchronize(stream);

#ifdef DEBUG_CUDA
	clock_t t1 = clock();
	er1 = hipEventCreate(&start);
	er1 = hipEventCreate(&stop);
	er1 = hipEventRecord(start, stream);
#endif

	// calculating nodes
	for (int i = 0; i < m_length_of_tree + 1; ++i) {
		er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
		if (atomic_iter > m_warp_size2 /*prop.warpSize * 2 */) {
			unsigned t = (unsigned)ceil((double)atomic_iter / (double)(m_warp_size2 /*prop.warpSize * 2.0 */));
			block_size = dim3(m_warp_size2 /*prop.warpSize * 2 */, 1, 1);
			grid_size = dim3(t, 1, 1);
		}
		else {
			grid_size = dim3(1, 1, 1);
			block_size = dim3(atomic_iter, 1, 1);
		}
		cuda_search_rect2_impl1 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_iter_count, dev_atomic_iter, dev_ptr2);

		er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
		er1 = hipMemcpyAsync(dev_ptr, dev_ptr2, sizeof(void*) * atomic_iter, hipMemcpyDeviceToDevice, stream);
		er1 = hipMemcpyAsync(dev_iter_count, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToDevice, stream);
		hipStreamSynchronize(stream);
		//printf("===== Iter %i: next = %u (%s)\n", i, atomic_iter, er1 == hipSuccess ? "true" : "false");
		//hipDeviceSynchronize();
	}
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(stop, stream);
	er1 = hipEventSynchronize(stop);
	er1 = hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 1 time = %f ms\n", gtime);
#endif

	// calculating branches
	grid_size = dim3(atomic_iter, 1, 1);
	block_size = dim3(m_warp_size2 /*prop.warpSize * 2 */, 1, 1);
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(start, stream);
#endif

	indexer *dev_idxs = NULL;
	hipMalloc((void**)&dev_idxs, sizeof(indexer) * atomic_iter);
	//hipMemsetAsync(dev_idxs, 0, sizeof(indexer), stream);
	coord *dev_dist = NULL;
	hipMalloc((void**)&dev_dist, sizeof(coord) * atomic_iter);
	//hipMemsetAsync(dev_dist, 0, sizeof(coord), stream);
	//er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	cuda_search_nearest_item2_impl2 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, /*dev_atomic_iter,*/ x, y, dev_idxs, dev_dist);
	//er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
	er1 = hipStreamSynchronize(stream);
	//er1 = hipDeviceSynchronize();
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(stop, stream);
	er1 = hipEventSynchronize(stop);
	er1 = hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 2 time = %f ms\n", gtime);
	clock_t t2 = clock();
	printf("All kernels time = %i ms\n", t2 - t1);
#endif

	grid_size = dim3(m_multi_processor_count, 1, 1), block_size = dim3(m_warp_size2 / 2, 1, 1);
	indexer *dev_idxs2 = NULL;
	hipMalloc((void**)&dev_idxs2, sizeof(indexer) * (size_t)ceil((double)atomic_iter / (double)m_warp_size2));
	coord *dev_dist2 = NULL;
	hipMalloc((void**)&dev_dist2, sizeof(coord) * (size_t)ceil((double)atomic_iter / (double)m_warp_size2));
	while (atomic_iter > 1) {
		er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
		cuda_search_nearest_item2_impl3 << <grid_size, block_size, 0, stream >> > (dev_idxs, dev_dist, atomic_iter, dev_atomic_iter, dev_idxs2, dev_dist2);
		er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
		er1 = hipMemcpyAsync(dev_idxs, dev_idxs2, sizeof(indexer) * atomic_iter, hipMemcpyDeviceToDevice, stream);
		er1 = hipMemcpyAsync(dev_dist, dev_dist2, sizeof(coord) * atomic_iter, hipMemcpyDeviceToDevice, stream);
		hipStreamSynchronize(stream);
	}

	er1 = hipMemcpyAsync(dist, dev_dist, sizeof(coord), hipMemcpyDeviceToHost, stream);
	indexer *idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * 1);
	//er1 = hipMemcpyAsync(idxs, host_idxs, sizeof(indexer) * *count_items, hipMemcpyHostToHost, stream);
	/*if (*dist == (coord)0.0)
		idxs[0] = (indexer)-1;
	else*/
		er1 = hipMemcpyAsync(idxs, dev_idxs, sizeof(indexer) * 1, hipMemcpyDeviceToHost, stream);
	er1 = hipStreamSynchronize(stream);
#ifdef DEBUG_CUDA
	printf("Total results from device = %u\n", 1);
#endif

	// freeing and destroying
	hipStreamDestroy(stream);

	er1 = hipFree(dev_iter_count);
	er1 = hipFree(dev_ptr);
	er1 = hipFree(dev_ptr2);
	//er1 = hipFree(dev_tmp_idxs);
	//er1 = hipFree(dev_count_items);
	er1 = hipFree(dev_atomic_iter);
	//er1 = hipHostFree(host_idxs);
	er1 = hipFree(dev_idxs);
#ifdef DEBUG_CUDA
	er1 = hipEventDestroy(stop);
	er1 = hipEventDestroy(start);
#endif

	return idxs;
}

/* searching the nearest item on device implementation */
__global__ void cuda_search_nearest_item2_impl2(void **br_ptr, /*indexer *atomic_iter,*/ coord x, coord y, /*out*/ indexer *idxs, /*out*/ coord *dist)
{
	int idxx = threadIdx.x;
	int idx_gr_br = blockIdx.x;

	// for store temporary results
	__shared__ indexer temp_res[65]; // must be as blockDim.x size + 1 (for rpevious result)
	//__shared__ char temp_res_flag[64];
	__shared__ coord curr_dist[65];
	
	//temp_res_flag[idxx] = -1;
	if (!idxx) {
		temp_res[64] = (indexer)-1;
		curr_dist[64] = FLT_MAX;
	}

	struct branch** br = (struct branch**)br_ptr;
	struct branch *curr_br = br[idx_gr_br];
	__syncthreads();

	__shared__ indexer start_num[1];
	if (!idxx)
		start_num[0] = curr_br->leaf_number[0];
	__syncthreads();

	if (curr_br->x_min <= dev_bonds->x_max && curr_br->x_max >= dev_bonds->x_min && curr_br->y_min <= dev_bonds->y_max && curr_br->y_max >= dev_bonds->y_min) {
		int t = (int)ceilf((float)curr_br->count_leafs / (float)blockDim.x);
		for (int j = 0; j < t; ++j) {
			curr_dist[idxx] = FLT_MAX;
			temp_res[idxx] = (indexer)-1;
			int curr_idx = idxx + j * blockDim.x; // curr_offset;
			if (/*j == t1 && */curr_idx < curr_br->count_leafs) {
				// loading frequantly using data
				__shared__ coord leaf_x[65];
				__shared__ coord leaf_y[65];
				leaf_x[idxx] = curr_br->leaf_x[curr_idx];
				leaf_y[idxx] = curr_br->leaf_y[curr_idx];
				if (!idxx && curr_idx + 64 < curr_br->count_leafs && curr_br->merge_next_leaf[curr_idx + 63]) {
					leaf_x[64] = curr_br->leaf_x[curr_idx + 64];
					leaf_y[64] = curr_br->leaf_y[curr_idx + 64];
				}

				// calculating distances
				if (curr_br->merge_next_leaf[curr_idx]) {
					curr_dist[idxx] = cuda_distance(x, y, leaf_x[idxx], leaf_y[idxx], leaf_x[idxx + 1], leaf_y[idxx + 1]);
				}
				else {
					indexer curr_num = curr_br->offset[curr_br->leaf_number[curr_idx] - start_num[0]];
					__shared__ coord leaf_x_offset[64];
					__shared__ coord leaf_y_offset[64];
					leaf_x_offset[idxx] = curr_br->leaf_x[curr_num];
					leaf_y_offset[idxx] = curr_br->leaf_y[curr_num];

					curr_dist[idxx] = cuda_distance(x, y, leaf_x[idxx], leaf_y[idxx], leaf_x_offset[idxx], leaf_y_offset[idxx]);
				}
				temp_res[idxx] = curr_br->leaf_number[curr_idx];
				__syncthreads();

				// find min distance
				for (int k = blockDim.x / 2; k > 0; k >>= 1) {
					if (idxx < k) {
						if (curr_dist[idxx] > curr_dist[idxx + k]) {
							curr_dist[idxx] = curr_dist[idxx + k];
							temp_res[idxx] = temp_res[idxx + k];
						}
					}
				}

				// check previous result
				if (!idxx) {
					if (curr_dist[64] > curr_dist[0]) {
						curr_dist[64] = curr_dist[0];
						temp_res[64] = temp_res[0];
					}
				}
				__syncthreads();
			}
		}
	}
	__syncthreads();
	if (!idxx) {
		dist[idx_gr_br] = curr_dist[64];
		idxs[idx_gr_br] = temp_res[64];
		//printf("GGGPU (%i) idx = %u, dist = %e\n", idx_gr_br, temp_res[64], curr_dist[64]);
	}
}

/* calculating distance between point and line */
__device__ coord cuda_distance(coord px, coord py, coord line_p0x, coord line_p0y, coord line_p1x, coord line_p1y)
{
	coord vx, vy, wx, wy, c1, c2, b, pbx, pby;

	vx = line_p1x - line_p0x;
	vy = line_p1y - line_p0y;
	wx = px - line_p0x;
	wy = py - line_p0y;
	c1 = vx * wx + vy * wy;

	if (c1 <= 0) {
		//coord t1 = p->x - line_p0->x;
		//coord t2 = p->y - line_p0->y;
		coord t1 = wx;
		coord t2 = wy;
		return (coord)sqrt(t1 * t1 + t2 * t2);
	}

	c2 = vx * vx + vy * vy;
	if (c2 <= c1) {
		//return sqrt(pow(fabs(p->x - line_p1->x), 2) + pow(fabs(p->y - line_p1->y), 2));
		coord t1 = px - line_p1x;
		coord t2 = py - line_p1y;
		return (coord)sqrt(t1 * t1 + t2 * t2);
	}

	b = c1 / c2;
	pbx = line_p0x + b * vx;
	pby = line_p0y + b * vy;

	//return sqrt(pow(fabs(p->x - pbx), 2) + pow(fabs(p->y - pby), 2));
	coord t1 = px - pbx;
	coord t2 = py - pby;
	return (coord)sqrt(t1 * t1 + t2 * t2);
}

/* searching the nearest item on device implementation (step 3) */
__global__ void cuda_search_nearest_item2_impl3(/*in*/ indexer *idxs, /*in*/ coord *dist, /*in*/indexer count, /*in*/indexer *atomic_iter, /*out*/ indexer *idxs2, /*out*/ coord *dist2)
{
	indexer idxx = threadIdx.x;

	//indexer thr_index = idxx + blockIdx.x * blockDim.x;
	//if (thr_index > count) {
	//	return;
	//}

	__shared__ coord d[64];
	d[idxx] = FLT_MAX;
	d[idxx + 32] = FLT_MAX;

	int block_size = gridDim.x * blockDim.x * 2;
	int c1 = (int)ceilf((double)count / (double)block_size);
	for (int i = 0; i < c1; ++i) {
		// find min distance
		indexer offset = blockIdx.x * blockDim.x * 2 + i * block_size;
		indexer curr_index = idxx + offset;
		//if (curr_index + 32 > count)
		//	break;
		if (curr_index < count)
			d[idxx] = dist[curr_index];
		if (curr_index + 32 < count)
			d[idxx + 32] = dist[curr_index + 32];
		__syncthreads();
		for (int k = blockDim.x /* * 2 / 2 */; k > 0; k >>= 1) {
			if (idxx < k) {
				//if (dist[curr_index] > dist[curr_index + k]) {
				if (d[idxx] > d[idxx + k]) {
					//dist[curr_index] = dist[curr_index + k];
					d[idxx] = d[idxx + k];
					idxs[curr_index] = idxs[curr_index + k];
				}
			}
			else {
				break;
			}
			__syncthreads();
			/*if (!idxx)
				printf("%i, k = %i, idx = %u:\n%e,%e,%e,%e,%e,%e,%e,%e\n%e,%e,%e,%e,%e,%e,%e,%e\n%e,%e,%e,%e,%e,%e,%e,%e\n%e,%e,%e,%e,%e,%e,%e,%e\n%e,%e,%e,%e,%e,%e,%e,%e\n%e,%e,%e,%e,%e,%e,%e,%e\n%e,%e,%e,%e,%e,%e,%e,%e\n%e,%e,%e,%e,%e,%e,%e,%e\n",
					blockIdx.x, k, curr_index,
					dist[curr_index], dist[curr_index + 1], dist[curr_index + 2], dist[curr_index + 3], dist[curr_index + 4], dist[curr_index + 5], dist[curr_index + 6], dist[curr_index + 7],
					dist[curr_index + 8], dist[curr_index + 9], dist[curr_index + 10], dist[curr_index + 11], dist[curr_index + 12], dist[curr_index + 13], dist[curr_index + 14], dist[curr_index + 15],
					dist[curr_index + 16], dist[curr_index + 17], dist[curr_index + 18], dist[curr_index + 19], dist[curr_index + 20], dist[curr_index + 21], dist[curr_index + 22], dist[curr_index + 23],
					dist[curr_index + 24], dist[curr_index + 25], dist[curr_index + 26], dist[curr_index + 27], dist[curr_index + 28], dist[curr_index + 29], dist[curr_index + 30], dist[curr_index + 31],
					dist[curr_index + 32], dist[curr_index + 33], dist[curr_index + 34], dist[curr_index + 35], dist[curr_index + 36], dist[curr_index + 37], dist[curr_index + 38], dist[curr_index + 39],
					dist[curr_index + 40], dist[curr_index + 41], dist[curr_index + 42], dist[curr_index + 43], dist[curr_index + 44], dist[curr_index + 45], dist[curr_index + 46], dist[curr_index + 47],
					dist[curr_index + 48], dist[curr_index + 49], dist[curr_index + 50], dist[curr_index + 51], dist[curr_index + 52], dist[curr_index + 53], dist[curr_index + 54], dist[curr_index + 55],
					dist[curr_index + 56], dist[curr_index + 57], dist[curr_index + 58], dist[curr_index + 59], dist[curr_index + 60], dist[curr_index + 61], dist[curr_index + 62], dist[curr_index + 63]);
		*/
		}
		__syncthreads();
		if (!idxx && curr_index < count) {
			int t = atomicAdd(atomic_iter, 1);
			idxs2[t] = idxs[offset];
			//dist2[t] = dist[offset];
			dist2[t] = d[0];
		}
	}
}