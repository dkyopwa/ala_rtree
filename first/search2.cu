#include "hip/hip_runtime.h"
/*

*/

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
//#include <math.h>
#include <float.h>
//#include <hip/hip_runtime.h>
#include "unimem.h"
#include "first.h"

#define DEBUG_CUDA_INFO
#define MAX_RESULTS 100000
#define PACK_RESULTS

struct boundaries {
	coord x_min;
	coord y_min;
	coord x_max;
	coord y_max;
	bool intersection;
};

struct node* m_dev_node = NULL;
__constant__ boundaries dev_bonds[1];
__constant__ unsigned dev_threads_count[1];
unsigned m_threads_count;
indexer m_count_branches;
int m_length_of_tree = 0;

//__constant__ struct branch *m_ttt_cuda_first_branch = NULL;

#ifdef PACK_RESULTS
/// compare
int cmp(const void* a, const void* b)
{
	return (int)(*(indexer*)a - *(indexer*)b);
}

#endif

extern "C"
bool init_cuda_device(int deviceID, struct node* node)
{
	if (!node)
		return false;
	//return false;

	struct node *nd = node;
	unsigned count1[64], i = 0;
	m_count_branches = 0;
	for (int j = 0; j < 64; j++) count1[j] = 0;
	//count1[0] = 1;

	struct node *stack_node[64];
	int stack_pos = 0;
	indexer stack_idx[64];
	alignas(16) struct branch *first_branch = NULL;
	alignas(16) struct node* stack_first_node[64];
	for (unsigned i = 0; i < 64; ++i) {
		stack_first_node[i] = NULL;
	}
	while (i < nd->count_child_nodes) {
		if (!stack_first_node[stack_pos] || nd < stack_first_node[stack_pos])
			stack_first_node[stack_pos] = nd;
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				if (!first_branch || br < first_branch)
					first_branch = br;
			}
			/*if (!count_br)
				count_br = nd->count_child_nodes;
			if (count_br != nd->count_child_nodes)
				printf("Branches %u vs %u\n", count_br, nd->count_child_nodes);*/
			m_count_branches += nd->count_child_nodes;
				// return from stack
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
				else {
					//if (count1[stack_pos]) {
						//if (count1[stack_pos] != nd->count_child_nodes) {
						//	printf("Nodes %u vs %u\n", count1[stack_pos], nd->count_child_nodes);
						//}
						count1[stack_pos] += nd->count_child_nodes;
					//}
					//else {
					//	count1[stack_pos] = nd->count_child_nodes;
					//}
				}
			}
		}
		else {
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			i = 0;
			nd = (struct node*)nd->child_node[i];
			/*if (!count1[stack_pos])
				count1[stack_pos] = nd->count_child_nodes;
			else
				count1[stack_pos] += nd->count_child_nodes;*/
		}
		/*} else if (i < nd->count_child_nodes) {
		i++;*/
		/*if (!count1[stack_pos])
			count1[stack_pos] = nd->count_child_nodes;
		else
			count1[stack_pos] += nd->count_child_nodes;
			*/
	}

	//return false;

	int deviceCount;
	hipError_t er1 = hipGetDeviceCount(&deviceCount);
	printf("DevicecheckCudaErrors Count: %d\n", deviceCount);

	if (deviceID == -1)
		deviceID = 0;

	hipDeviceProp_t prop;
	for (int ii = 0; ii < deviceCount; ++ii) {
		er1 = hipGetDeviceProperties(&prop, ii);
		if (prop.major < 2 || prop.canMapHostMemory != 1)
		{
			printf("ERROR: calculation requires GPU devices with compute SM 2.0 or higher, or can not using MapHostMemory.\n");
			printf("Current GPU device has compute SM%d.%d, Exiting...", prop.major, prop.minor);
			//exit(EXIT_WAIVED);
			return false;
		}

		printf("GPU device name is %s\n", prop.name);
		printf("GPU total memory = %.0f Mb\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		printf("Number of multiprocessors on the device = %u\n", prop.multiProcessorCount);
	}

	er1 = hipSetDevice(deviceID);
	hipSetDeviceFlags(hipDeviceMapHost);
	er1 = hipGetDeviceProperties(&prop, deviceID);
	m_threads_count = prop.multiProcessorCount * prop.warpSize;
	//er1 = hipMalloc((void**)&dev_threads_count, sizeof(unsigned));
	er1 = hipMemcpyToSymbol(HIP_SYMBOL(dev_threads_count), &m_threads_count, sizeof(unsigned));

	// copy rtree
	int pos = 63;

	for (; pos >= 0; --pos) {
		if (count1[pos])
			break;
	}
	m_length_of_tree = pos + 1;

	// allocationg memory for branches
	alignas(16) struct branch* tbr = (struct branch*)aligned_alloc(16, sizeof(struct branch) * m_count_branches);
	// struct branch* first_branch = NULL;
	/*nd = node;
	i = 0;
	unsigned k = 0;
	while (i < nd->count_child_nodes) {
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				//if (!first_branch || br < first_branch)
				//	first_branch = br;
				memcpy(tbr + k, br, sizeof(struct branch));
				k++;
			}
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					// insert to stack
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
			}
		}
		else {
			// insert to stack
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			nd = (struct node*)nd->child_node[i];
		}
	}*/
	/*nd = node;
	i = 0;
	for (int j = 0; j <= pos; ++j)
		nd = (struct node*)(nd->child_node[0]);
	for (indexer j = 0; j < count1[pos]; ++j) {
		for (indexer k = 0; k < nd[j].count_child_nodes; ++k) {
			memcpy(tbr + i, nd[j].child_node[k], sizeof(struct branch));
			i++;
		}
	}
	*/
	memcpy(tbr, first_branch, sizeof(struct branch) * m_count_branches);

	// for debug
	/*printf("\n\n\n======================================================================\n");
	for (indexer i = 0; i < count1[pos]; ++i) {
		if ((struct node*)(stack_first_node[pos + 1])[i].is_last_node) {
			unsigned tt = ((struct node*)(stack_first_node[pos + 1]))[i].count_child_nodes;
			for (indexer ii = 0; ii < tt; ++ii) {
				unsigned idx = (struct branch*)((struct node*)(stack_first_node[pos + 1])[i].child_node[ii]) - first_branch;
				if (!idx)
					printf("0\n");
				else
					printf("%u\n", idx);
			}
		}
		else {
			printf("Error last node %u\n", i);
		}
	}*/

	// copy data of branches to device
	clock_t t1 = clock();
	hipStream_t stream;
	hipStreamCreate(&stream);
	for (indexer i = 0; i < m_count_branches; ++i)
	{
		void *data_ptr = tbr[i].leaf_x;
		er1 = hipMalloc((void**)&(tbr[i].leaf_x), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_x, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_y;
		er1 = hipMalloc((void**)&(tbr[i].leaf_y), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_y, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_number;
		er1 = hipMalloc((void**)&(tbr[i].leaf_number), sizeof(indexer) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_number, data_ptr, sizeof(indexer) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].merge_next_leaf;
		er1 = hipMalloc((void**)&(tbr[i].merge_next_leaf), sizeof(bool) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].merge_next_leaf, data_ptr, sizeof(bool) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		/*data_ptr = tbr[i].xsh_min;
		er1 = hipMalloc((void**)&(tbr[i].xsh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].xsh_max;
		er1 = hipMalloc((void**)&(tbr[i].xsh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_min;
		er1 = hipMalloc((void**)&(tbr[i].ysh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_max;
		er1 = hipMalloc((void**)&(tbr[i].ysh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream); */
		data_ptr = tbr[i].offset;
		er1 = hipMalloc((void**)&(tbr[i].offset), sizeof(indexer) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].offset, data_ptr, sizeof(indexer) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
	}
	er1 = hipStreamSynchronize(stream);
	er1 = hipStreamDestroy(stream);
	clock_t t2 = clock();
	printf("Time copying data to device = %u ms\n", t2 - t1);

	// copy branches to device
	struct branch *dev_br = NULL;
	er1 = hipMalloc((void**)&dev_br, sizeof(struct branch) * m_count_branches);
	er1 = hipMemcpy(dev_br, tbr, sizeof(struct branch) * m_count_branches, hipMemcpyHostToDevice);
	//hipMemcpyToSymbol(HIP_SYMBOL(m_ttt_cuda_first_branch), &dev_br, sizeof(struct branch*));

	//return false;
	alignas(16) struct node *to_dev_nd[65];
	//void **to_dev_child[64];
	struct node *dev_nd = NULL, *dev_nd_prev = NULL, *dev_ptr = NULL;
	// to_dev_nd[0] = (struct node*)aligned_alloc(16, sizeof(struct node));
	// memcpy(to_dev_nd[0], nd, sizeof(struct node));
	struct node* tnd = node;
	//for (unsigned j = 0; j <= pos; ++j)
	//	tnd = (struct node*)(tnd->child_node[0]);
	//unsigned j = 0;
	//void* tmp1 = NULL;
	unsigned count = tnd->count_child_nodes, prev_count = 1;
	//printf("\n\n\n======================================================================\n");
	for (int k1 = pos; k1 >= 0; --k1) {
		tnd = node;
		//for (unsigned j = 0; j <= k1; ++j)
		//	tnd = (struct node*)(tnd->child_node[0]);
		// data child node
		to_dev_nd[k1] = (struct node*)aligned_alloc(16, sizeof(struct node) * count1[k1]);
		//memcpy(to_dev_nd[k1], tnd/*->child_node[0]*/, sizeof(struct node) * count1[k1]);
		memcpy(to_dev_nd[k1], stack_first_node[k1 + 1], sizeof(struct node) * count1[k1]);
		// pointer to child_node on host
		for (indexer j = 0; j < count1[k1]; ++j) {
			//(to_dev_nd[k1])[j]->child_node = (void**)aligned_alloc(16, sizeof(void*) * MAX_NODES); // tnd->count_child_nodes);
			//(to_dev_child[k1])[j] = to_dev_nd[j]->child_node;
			dev_ptr = NULL;
			er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * MAX_NODES);
			(to_dev_nd[k1])[j].child_node = (void**)dev_ptr;
			for (indexer k2 = 0; k2 < MAX_NODES; ++k2) {
				if (k1 == pos) {
					// copy pointer of branches
					//struct branch *ptr = &(dev_br[k2 + j * MAX_NODES]);
					unsigned idx = (struct branch*)((struct node*)(stack_first_node[k1 + 1])[j].child_node[k2]) - first_branch;
					//if (idx == 4899)
					//	printf("%u\n", idx);
					struct branch *ptr = &(dev_br[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct branch*), hipMemcpyHostToDevice);
				}
				else {
					// copy pointer of nodes
					//struct node* ptr = &(dev_nd_prev[k2 + j * MAX_NODES]);
					unsigned idx = (struct node*)(stack_first_node[k1 + 1])[j].child_node[k2] - (struct node*)(stack_first_node[k1 + 2]);
					//printf("%u\n", idx);
					struct node *ptr = &(dev_nd_prev[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
				}
			}
		}
		//printf("==========================================\n\n\n");
		// pointers of child nodes
		er1 = hipMalloc((void**)&dev_nd, sizeof(struct node) * count1[k1]); // tnd->count_child_nodes);
		hipMemcpy(dev_nd, to_dev_nd[k1], sizeof(struct node) * count1[k1], hipMemcpyHostToDevice);
		dev_nd_prev = dev_nd;
	}
	// copy top node (root)
	to_dev_nd[64] = (struct node*)aligned_alloc(16, sizeof(struct node));
	memcpy(to_dev_nd[64], node/*->child_node[0]*/, sizeof(struct node));
	dev_ptr = NULL;
	er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * node->count_child_nodes);
	(to_dev_nd[64])[0].child_node = (void**)dev_ptr;
	for (indexer k2 = 0; k2 < node->count_child_nodes; ++k2) {
		// copy pointer of nodes
		//struct node* ptr = &(dev_nd_prev[k2]);
		unsigned idx = (struct node*)(stack_first_node[0])[0].child_node[k2] - (struct node*)(stack_first_node[1]);
		struct node* ptr = &(dev_nd_prev[idx]);
		er1 = hipMemcpy((void*)((to_dev_nd[64])[0].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
	}
	// pointers of child nodes
	er1 = hipMalloc((void**)&dev_nd, sizeof(struct node)); // tnd->count_child_nodes);
	er1 = hipMemcpy(dev_nd, to_dev_nd[64], sizeof(struct node), hipMemcpyHostToDevice);
	m_dev_node = dev_nd;
	printf("============== 0x%llx, 0x%llx, prev = 0x%llx\n", m_dev_node, dev_nd, dev_nd_prev);

	// free memory
	for (int k1 = pos; k1 >= 0; --k1) {
		_aligned_free(to_dev_nd[k1]);
	}
	_aligned_free(to_dev_nd[64]);

	// allocating memory for root
	//er1 = hipMalloc((void**)&m_dev_node, sizeof(struct node));
	// copy to device root of tree
	//hipMemcpy(m_dev_node, to_dev_nd[0], sizeof(struct node), hipMemcpyHostToDevice);

	return true;
}

extern "C"
bool destroy_cuda_device()
{
	//hipFree(dev_threads_count);
	hipError_t er1 = hipDeviceReset();
	return er1 == hipSuccess ? true : false;
}

extern "C"
#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
indexer* cuda_search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items, ret_callback2_circle callback = NULL, void *data = NULL);
/* searchin items in selected rectangle on cuda device imlementation */
__global__ void cuda_search_rect2_impl1(void **nd, indexer *iter_count, indexer *atomic_iter, /*out*/ void **next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items, ret_callback2_circle callback = NULL, void *data = NULL);
__global__ void cuda_search_rect2_impl2(void **nd, int *iter_count, indexer *atomic_iter, /*out*/ void **next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items);
__global__ void cuda_search_rect2_impl3(indexer *unpack_idxs, indexer *iter_count, indexer *atomic_iter, /*out*/ void **next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items);
#else
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
__global__ indexer* search_rect2_impl(void *nd_ptr, indexer iter_count, /*out*/indexer *count_items)
#endif // CALC_POINT

#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
indexer* cuda_search_rect2(node * nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, indexer * count_items, ret_callback2_circle callback, void * data)
#else
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
#endif // CALC_POINT
{

	// memory for result
	/*size_t mem_size = 256;
	size_t count_mem = 1;
	alignas(16) indexer* idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * mem_size * count_mem); */
	hipError_t er1;
	hipStream_t stream;
	hipStreamCreate(&stream);

	indexer *host_idxs = NULL, *dev_idxs = NULL, *dev_tmp_idxs = NULL;;
	hipHostAlloc((void**)&host_idxs, sizeof(indexer) * MAX_RESULTS, hipHostMallocMapped);
	hipHostGetDevicePointer((void**)&dev_idxs, host_idxs, 0);
	hipMalloc((void**)&dev_tmp_idxs, sizeof(indexer) * MAX_RESULTS);

	// searching
	hipEvent_t start, stop;
	float gtime = 0.0;
	int device_id;
	hipDeviceProp_t prop;
	er1 = hipGetDevice(&device_id);
	er1 = hipGetDeviceProperties(&prop, device_id);
	dim3 grid_size = dim3(prop.multiProcessorCount, 1, 1), block_size = dim3(prop.warpSize, 1, 1);
	// store boundaries
	boundaries b1;
	b1.intersection = intersection; b1.x_max = x_max; b1.x_min = x_min; b1.y_max = y_max; b1.y_min = y_min;
	//hipMalloc((void**)dev_bonds, sizeof(struct boundaries));
	hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_bonds), &b1, sizeof(struct boundaries), 0, hipMemcpyHostToDevice, stream);
	// for store count of iterations to next step
	indexer *dev_atomic_iter = NULL;
	hipMalloc((void**)&dev_atomic_iter, sizeof(indexer));
	hipMemsetAsync(dev_atomic_iter, 0, 1, stream);
	// store pointers for next step
	void **dev_ptr = NULL, **dev_ptr2 = NULL;
	hipMalloc((void**)&dev_ptr, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx, count_br = %u\n", &m_dev_node, m_dev_node, m_count_branches);
	void **tptr = (void**)(&m_dev_node);
	hipMemcpyAsync(dev_ptr, tptr, sizeof(void*), hipMemcpyHostToDevice, stream);
	hipMalloc((void**)&dev_ptr2, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx; dev_ptr = 0x%llx\n", &m_dev_node, m_dev_node, dev_ptr);
	// count items
	indexer *dev_count_items = NULL;
	hipMalloc((void**)&dev_count_items, sizeof(indexer));
	// count of iterations
	indexer *dev_iter_count = NULL;
	hipMalloc((void**)&dev_iter_count, sizeof(indexer));
	hipMemsetAsync(dev_iter_count, 1, 1, stream);
	hipStreamSynchronize(stream);

	indexer atomic_iter = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, stream);
	
	// calculating nodes
	for (int i = 0; i < m_length_of_tree + 1; ++i) {
		er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
		cuda_search_rect2_impl1 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_iter_count, dev_atomic_iter, dev_ptr2, dev_idxs, count_items, callback, data);

		er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
		er1 = hipMemcpyAsync(dev_ptr, dev_ptr2, sizeof(void*) * atomic_iter, hipMemcpyDeviceToDevice, stream);
		er1 = hipMemcpyAsync(dev_iter_count, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToDevice, stream);
		hipStreamSynchronize(stream);
		printf("===== Iter %i: next = %u (%s)\n", i, atomic_iter, er1 == hipSuccess ? "true" : "false");
		//hipDeviceSynchronize();
	}

	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 1 time = %f ms\n", gtime);

	// calculating branches
	grid_size = dim3(atomic_iter, 1, 1);
	hipEventRecord(start, stream);
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	cuda_search_rect2_impl2 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, NULL, dev_atomic_iter, dev_ptr2, dev_idxs, count_items);
	er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	hipDeviceSynchronize();
	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 2 time = %f ms\n", gtime);

	/*grid_size = dim3(prop.multiProcessorCount, 1, 1);
	hipEventRecord(start, stream);
	er1 = hipMemcpyAsync(dev_iter_count, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToDevice, stream);
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	cuda_search_rect2_impl3 << <grid_size, block_size, 0, stream >> > (dev_tmp_idxs, dev_iter_count, dev_atomic_iter, dev_ptr2, dev_idxs, count_items);
	hipStreamSynchronize(stream);
	hipDeviceSynchronize();
	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 3 time = %f ms\n", gtime);*/

	hipMemcpyAsync(count_items, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);

	indexer *idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * *count_items);
	er1 = hipMemcpyAsync(idxs, host_idxs, sizeof(indexer) * *count_items, hipMemcpyHostToHost, stream);
	hipStreamSynchronize(stream);

	// freeing and destroying
	hipStreamDestroy(stream);

	er1 = hipFree(dev_iter_count);
	er1 = hipFree(dev_ptr);
	er1 = hipFree(dev_ptr2);
	er1 = hipFree(dev_tmp_idxs);
	er1 = hipFree(dev_count_items);
	er1 = hipFree(dev_atomic_iter);
	hipEventDestroy(stop);
	hipEventDestroy(start);
	hipHostFree(host_idxs);

#ifdef PACK_RESULTS
	if (*count_items) {
		qsort(idxs, *count_items, sizeof(indexer), cmp);
		indexer j = 1;
		indexer offset = 0;
		for (indexer i = 0; i < *count_items - 1 - offset; ++i) {
			if (idxs[i] == idxs[i + 1 + offset]) {
				offset++;
				idxs[i + 1] = idxs[i + 1 + offset];
			}
			if (offset)
				idxs[i + 1] = idxs[i + 1 + offset];
			j++;
		}
		*count_items = j;
		idxs = (indexer*)_aligned_realloc(idxs, sizeof(indexer) * j, 16);
	}
#endif
	return idxs;
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl1(void **nd_ptr, indexer *iter_count, indexer *atomic_iter, /*out*/ void** next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items, ret_callback2_circle callback, void *data)
{
	int idxx = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("Thread = %u\n", idxx);
	//int idxx_t = idxx % (*dev_threads_count);
	indexer iter_count_t = *iter_count % (*dev_threads_count);
	if (!iter_count_t)
		iter_count_t = *iter_count;

	//if (!idxx)
	//	printf("Calc NODE iter_count = %u (0x%llx)\n", *iter_count, iter_count);

	struct node** nd = (struct node**)nd_ptr;
	//indexer idx = 0;

#ifdef CALC_POINT
	//coord tmp_dist = FLT_MAX;
	//indexer tmp_idx = -1;
#endif // CALC_POINT
	int t = (int)ceilf((float)*iter_count / (float)*dev_threads_count);
	int t1 = t - 1;
	for (int j = 0; j < t; ++j) {
		//printf("Thread = %i, j = %u, t1 = %u, idxx = %u, >= iter_count_t = %u (%s => %s)\n", idxx, j, t1, idxx, iter_count_t, idxx >= iter_count_t ? "true" : "false", j == t1 && idxx >= iter_count_t ? "true" : "false");
		if (j == t1 && idxx >= iter_count_t) {// idxx_t >= *iter_count) {
			return;
		}
		//printf("Thread %i (%i): x1 = %f, x2 = %f, y1 = %f, y2 = %f\n", idxx + j * (*dev_threads_count), j, nd[idxx + j * (*dev_threads_count)]->x1, nd[idxx + j * (*dev_threads_count)]->x2, nd[idxx + j * (*dev_threads_count)]->y1, nd[idxx + j * (*dev_threads_count)]->y2);
		// node in bounrary or bounrary in node
		if (nd[idxx + j * (*dev_threads_count)]->x1 <= dev_bonds->x_max && nd[idxx + j * (*dev_threads_count)]->x2 >= dev_bonds->x_min && nd[idxx + j * (*dev_threads_count)]->y1 <= dev_bonds->y_max && nd[idxx + j * (*dev_threads_count)]->y2 >= dev_bonds->y_min) {
			//printf("Thread %i (%i) ================================ ====\n", idxx + j * (*dev_threads_count), j);
			// check node fully in the boundary
			//if (nd[idxx].x1 >= dev_bonds->x_min && nd[idxx].y1 >= dev_bonds->y_min && nd[idxx].x2 <= dev_bonds->x_max && nd[idxx].y2 <= dev_bonds->y_max) {
				// node is fully in the boundary
			//}
			//else {
				// node isn't fully in the boundary, than add to calculation to next iteration
				indexer t1 = atomicAdd(atomic_iter, nd[idxx + j * (*dev_threads_count)]->count_child_nodes);
				//printf("Increase %i: %u to %u (%u)\n", idxx, t1, *atomic_iter, nd[idxx]->count_child_nodes);
				/*if (t1 + nd->count_child_nodes >= 10000)
					return; */
				for (unsigned k = t1, t2 = 0; k < t1 + nd[idxx + j * (*dev_threads_count)]->count_child_nodes; ++k, ++t2) {
					//void **ptr = &next_nd;
					next_nd[k] = nd[idxx + j * (*dev_threads_count)]->child_node[t2];
					//printf("Next index = %u\n", (struct branch*)(nd[idxx + j * (*dev_threads_count)]->child_node[t2]) - m_ttt_cuda_first_branch);
				}
			//}
		}
		else {
			// node and boundary isn't intersection
		}
	}
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl2(void **br_ptr, int *iter_count, indexer *atomic_iter, /*out*/ void** next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items)
{
	int idxx = threadIdx.x;
	int idx_gr_br = blockIdx.x;
	//printf("Thread = %u\n", idxx);
	//if (!idxx /*&& !idx_gr_br*/)
	//	printf("============================================== block = %d, %u\n", idx_gr_br, *atomic_iter);

	// for store temporary results
	__shared__ indexer temp_res[33]; // must be as blockDim.x size + 1 (for rpevious result)
	//__shared__ indexer temp_res2[32];
	__shared__ char temp_res_flag[32];
	//__shared__ int atom_index[1];
	temp_res[idxx] = (indexer)-1;
	//temp_res2[idxx] = (indexer)-1;
	if (!idxx)
		temp_res[32] = (indexer)-1;
	temp_res_flag[idxx] = -1;
	//__shared__ int index = 0;

	//if (!idxx)
	//	printf("Calc BR iter_count = %u (0x%llx), grid.x = %i\n", *iter_count, iter_count, idx_gr_br);

	//struct branch* br = (struct branch*)br_ptr[0];
	struct branch** br = (struct branch**)br_ptr;
	//if (!idxx)
	//printf("Thread branch %i: x1 = %f, x2 = %f, y1 = %f, y2 = %f (u)\n", idx_gr_br, br[idx_gr_br]->x_min, br[idx_gr_br]->x_max, br[idx_gr_br]->y_min, br[idx_gr_br]->y_max); // , br[idx_gr_br] - m_ttt_cuda_first_branch);

	if (br[idx_gr_br]->x_min <= dev_bonds->x_max && br[idx_gr_br]->x_max >= dev_bonds->x_min && br[idx_gr_br]->y_min <= dev_bonds->y_max && br[idx_gr_br]->y_max >= dev_bonds->y_min) {
		//if (!idxx)
		//	printf("------------------ %i, %u\n", idx_gr_br, br[idx_gr_br]->count_shapes);
		//int idxx_t = idxx % blockDim.x;
		int t = (int)ceilf((float)br[idx_gr_br]->count_leafs / (float)blockDim.x);
		int t1 = t - 1;
		for (int j = 0; j < t; ++j) {
			int curr_offset = j * blockDim.x;
			if (j == t1 && idxx + curr_offset >= br[idx_gr_br]->count_leafs) {
				break;
			}

			// check points to enter to boundary
			if (br[idx_gr_br]->leaf_x[idxx + curr_offset] >= dev_bonds->x_min && br[idx_gr_br]->leaf_x[idxx + curr_offset] <= dev_bonds->x_max && br[idx_gr_br]->leaf_y[idxx + curr_offset] >= dev_bonds->y_min && br[idx_gr_br]->leaf_y[idxx + curr_offset] <= dev_bonds->y_max) {
				temp_res[idxx] = br[idx_gr_br]->leaf_number[idxx + curr_offset];
				/*int t2 = atomicAdd(atomic_iter, 1);
				if (t2 >= MAX_RESULTS - 1) {
					// can not store result
					atomicSub(atomic_iter, 1);
				}
				else {
					// can store result
					//idxs[t2] = 
				}*/
			}
			//if (!idxx)
			//	atom_index[0] = 0;
			__syncthreads();

			// fill empty places
			/*if (temp_res2[idxx] != (indexer)-1) {
				int t2 = atomicAdd(atom_index, 1);
				temp_res[t2] = temp_res2[idxx];
				printf("Temp (%i): %i => %i, res = %u (%u)\n", idxx, t2, atom_index[0], temp_res[t2], temp_res2[idxx]);
				if (t2 >= 32)
					printf("Vai vai vai %i => %i\n", t2, atom_index[0]);
			}
			//__threadfence();
			__syncthreads();
			if (!idxx)
			printf("1 === %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i\n",
				temp_res[0], temp_res[1], temp_res[2], temp_res[3], temp_res[4], temp_res[5], temp_res[6], temp_res[7], temp_res[8], temp_res[9],
				temp_res[10], temp_res[11], temp_res[12], temp_res[13], temp_res[14], temp_res[15], temp_res[16], temp_res[17], temp_res[18], temp_res[19],
				temp_res[20], temp_res[21], temp_res[22], temp_res[23], temp_res[24], temp_res[25], temp_res[26], temp_res[27], temp_res[28], temp_res[29],
				temp_res[30], temp_res[31]);*/

			// packing temporary results
			if (temp_res[idxx] == temp_res[idxx + 1]) {
				__threadfence();
				temp_res[idxx + 1] = -1;
				//__threadfence();
				//if (temp_res[idxx] != -1)
				//	printf("Index of equals items = %u + %u => %u\n", idxx, idxx + 1, temp_res[idxx + 1]);
			}
			else {
				//__threadfence();
			}
			__syncthreads();
			/*if (!idxx)
			printf("2 === %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i\n",
				temp_res[0], temp_res[1], temp_res[2], temp_res[3], temp_res[4], temp_res[5], temp_res[6], temp_res[7], temp_res[8], temp_res[9],
				temp_res[10], temp_res[11], temp_res[12], temp_res[13], temp_res[14], temp_res[15], temp_res[16], temp_res[17], temp_res[18], temp_res[19],
				temp_res[20], temp_res[21], temp_res[22], temp_res[23], temp_res[24], temp_res[25], temp_res[26], temp_res[27], temp_res[28], temp_res[29],
				temp_res[30], temp_res[31]);*/
			// ckeck for previous result
			if (!idxx) {
				//if (temp_res[0] != (unsigned)-1) printf("%i:%i Prev step = %u (%u)\n", idx_gr_br, j, temp_res[32], temp_res[0]);
				if (temp_res[32] == temp_res[0]) {
					//if (temp_res[0] != (unsigned)-1) printf("From prev step = %u\n", temp_res[0]);
					temp_res[0] = -1;
				}
			}
			//__syncthreads();

			// store temporary results to global array2
			if (temp_res[idxx] != -1) {
				int t2 = atomicAdd(atomic_iter, 1);
				if (t2 >= MAX_RESULTS - 1) {
					// can not store result
					atomicSub(atomic_iter, 1);
				}
				else {
					// can store result (idxs2 - temporary)
					idxs[t2] = temp_res[idxx];
					//printf("Number = %u, value = %u (%u)\n", t2, temp_res[idxx], idxs[t2]);
					temp_res_flag[idxx] = idxx;
					//printf("Increase %i: %u to %u\n", idxx, t2, *iter_count);
				}
			}

			__syncthreads();

			// store previous result
			for (int t2 = blockDim.x / 2; t2 > 0; t2 >>= 1)
			{
				if (idxx < t2) {
					if (temp_res_flag[idxx] < temp_res_flag[idxx + t2])
						temp_res_flag[idxx] = temp_res_flag[idxx + t2];
				}
				__syncthreads();
			}
			if (!idxx) {
				if (temp_res_flag[idxx] != -1) {
					//printf("INDEX === %u\n", temp_res_flag[idxx]);
					temp_res[32] = temp_res[temp_res_flag[idxx]]; // idxs[idxx];
				}
				//if (temp_res[32] != (unsigned)-1) printf("%i:%i Next step = %u\n", idx_gr_br, j, temp_res[32]);
			}
			
			// reset temporary resulats
			temp_res[idxx] = -1;
			//temp_res2[idxx] = -1;
			temp_res_flag[idxx] = -1;
			__syncthreads();
		}
	}
}

__global__ void cuda_search_rect2_impl3(indexer *unpack_idxs, indexer *iter_count, indexer *atomic_iter, /*out*/ void **next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items)
{
	int idxx = threadIdx.x + blockIdx.x * blockDim.x;

	// for store temporary results
	__shared__ indexer temp_res[33]; // must be as blockDim.x size + 1 (for rpevious result)
	__shared__ char temp_res_flag[32];
	temp_res[idxx] = (indexer)-1;
	if (!idxx)
		temp_res[32] = (indexer)-1;
	temp_res_flag[idxx] = -1;

	int t = (int)ceilf((float)*iter_count / (float)blockDim.x);
	int t1 = t - 1;
	temp_res[idxx] = -1;
	temp_res_flag[idxx] = -1;
	if (!idxx) {
		temp_res[32] = -1;
		printf("---------------------- Count temp results = %u (t = %i) ------------------\n", *iter_count, t);
	}

	for (int j = 0; j < t; ++j) {
		int curr_offset = j * blockDim.x;
		if (j == t1 && idxx + curr_offset >= *iter_count) {
			break;
		}
		temp_res[idxx] = unpack_idxs[idxx + curr_offset];

		if (temp_res[idxx] == temp_res[idxx + 1]) {
			__threadfence();
			temp_res[idxx + 1] = -1;
		}
		__syncthreads();

		if (!idxx) {
			if (temp_res[32] == temp_res[0]) {
				temp_res[0] = -1;
			}
		}

		//__threadfence();
		if (temp_res[idxx] == 2501949) {
			printf("1 =========================================================== 250 ======================================== %i, j = %i\n", idxx, j);
			printf("1 === %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i, %i\n",
				temp_res[0], temp_res[1], temp_res[2], temp_res[3], temp_res[4], temp_res[5], temp_res[6], temp_res[7], temp_res[8], temp_res[9],
				temp_res[10], temp_res[11], temp_res[12], temp_res[13], temp_res[14], temp_res[15], temp_res[16], temp_res[17], temp_res[18], temp_res[19],
				temp_res[20], temp_res[21], temp_res[22], temp_res[23], temp_res[24], temp_res[25], temp_res[26], temp_res[27], temp_res[28], temp_res[29],
				temp_res[30], temp_res[31]);
		}

		// store results to global array
		if (temp_res[idxx] != -1) {
			int t2 = atomicAdd(atomic_iter, 1);
			if (t2 >= MAX_RESULTS - 1) {
				// can not store result
				atomicSub(atomic_iter, 1);
			}
			else {
				// can store result (idxs2 - temporary)
				idxs[t2] = temp_res[idxx];
				//printf("Number = %u, value = %u (%u)\n", t2, temp_res[idxx], idxs[t2]);
				temp_res_flag[idxx] = idxx;
				//printf("Increase %i: %u to %u\n", idxx, t2, *iter_count);
			}
		}
		// store previous result
		for (int t2 = blockDim.x / 2; t2 > 0; t2 >>= 1)
		{
			if (idxx < t2) {
				if (temp_res_flag[idxx] < temp_res_flag[idxx + t2])
					temp_res_flag[idxx] = temp_res_flag[idxx + t2];
			}
			__syncthreads();
		}
		if (!idxx) {
			if (temp_res_flag[idxx] != -1) {
				//printf("INDEX === %u\n", temp_res_flag[idxx]);
				temp_res[32] = temp_res[temp_res_flag[idxx]]; // idxs[idxx];
			}
			//if (temp_res[32] != (unsigned)-1) printf("%i:%i Next step = %u\n", idx_gr_br, j, temp_res[32]);
		}
	}
}
