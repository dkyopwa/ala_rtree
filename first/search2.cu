#include "hip/hip_runtime.h"
/*

*/

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
//#include <math.h>
#include <float.h>
//#include <hip/hip_runtime.h>
#include "unimem.h"
#include "first.h"

#define DEBUG_CUDA_INFO
#define MAX_RESULTS 100000

struct boundaries {
	coord x_min;
	coord y_min;
	coord x_max;
	coord y_max;
	bool intersection;
};

struct node* m_dev_node = NULL;
__constant__ boundaries dev_bonds[1];
__constant__ unsigned dev_threads_count[1];
unsigned m_threads_count;
indexer m_count_branches;
int m_length_of_tree = 0;

extern "C"
bool init_cuda_device(int deviceID, struct node* node)
{
	if (!node)
		return false;
	//return false;

	struct node *nd = node;
	unsigned count1[64], i = 0;
	m_count_branches = 0;
	for (int j = 0; j < 64; j++) count1[j] = 0;
	//count1[0] = 1;

	struct node *stack_node[64];
	int stack_pos = 0;
	indexer stack_idx[64];
	alignas(16) struct branch *first_branch = NULL;
	alignas(16) struct node* stack_first_node[64];
	for (unsigned i = 0; i < 64; ++i) {
		stack_first_node[i] = NULL;
	}
	while (i < nd->count_child_nodes) {
		if (!stack_first_node[stack_pos] || nd < stack_first_node[stack_pos])
			stack_first_node[stack_pos] = nd;
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				if (!first_branch || br < first_branch)
					first_branch = br;
			}
			/*if (!count_br)
				count_br = nd->count_child_nodes;
			if (count_br != nd->count_child_nodes)
				printf("Branches %u vs %u\n", count_br, nd->count_child_nodes);*/
			m_count_branches += nd->count_child_nodes;
				// return from stack
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
				else {
					//if (count1[stack_pos]) {
						//if (count1[stack_pos] != nd->count_child_nodes) {
						//	printf("Nodes %u vs %u\n", count1[stack_pos], nd->count_child_nodes);
						//}
						count1[stack_pos] += nd->count_child_nodes;
					//}
					//else {
					//	count1[stack_pos] = nd->count_child_nodes;
					//}
				}
			}
		}
		else {
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			i = 0;
			nd = (struct node*)nd->child_node[i];
			/*if (!count1[stack_pos])
				count1[stack_pos] = nd->count_child_nodes;
			else
				count1[stack_pos] += nd->count_child_nodes;*/
		}
		/*} else if (i < nd->count_child_nodes) {
		i++;*/
		/*if (!count1[stack_pos])
			count1[stack_pos] = nd->count_child_nodes;
		else
			count1[stack_pos] += nd->count_child_nodes;
			*/
	}

	//return false;

	int deviceCount;
	hipError_t er1 = hipGetDeviceCount(&deviceCount);
	printf("DevicecheckCudaErrors Count: %d\n", deviceCount);

	if (deviceID == -1)
		deviceID = 0;

	hipDeviceProp_t prop;
	for (int ii = 0; ii < deviceCount; ++ii) {
		er1 = hipGetDeviceProperties(&prop, ii);
		if (prop.major < 2 || prop.canMapHostMemory != 1)
		{
			printf("ERROR: calculation requires GPU devices with compute SM 2.0 or higher, or can not using MapHostMemory.\n");
			printf("Current GPU device has compute SM%d.%d, Exiting...", prop.major, prop.minor);
			//exit(EXIT_WAIVED);
			return false;
		}

		printf("GPU device name is %s\n", prop.name);
		printf("GPU total memory = %.0f Mb\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		printf("Number of multiprocessors on the device = %u\n", prop.multiProcessorCount);
	}

	er1 = hipSetDevice(deviceID);
	hipSetDeviceFlags(hipDeviceMapHost);
	er1 = hipGetDeviceProperties(&prop, deviceID);
	m_threads_count = prop.multiProcessorCount * prop.warpSize;
	//er1 = hipMalloc((void**)&dev_threads_count, sizeof(unsigned));
	er1 = hipMemcpyToSymbol(HIP_SYMBOL(dev_threads_count), &m_threads_count, sizeof(unsigned));

	// copy rtree
	int pos = 63;

	for (; pos >= 0; --pos) {
		if (count1[pos])
			break;
	}
	m_length_of_tree = pos + 1;

	// allocationg memory for branches
	alignas(16) struct branch* tbr = (struct branch*)aligned_alloc(16, sizeof(struct branch) * m_count_branches);
	// struct branch* first_branch = NULL;
	/*nd = node;
	i = 0;
	unsigned k = 0;
	while (i < nd->count_child_nodes) {
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				//if (!first_branch || br < first_branch)
				//	first_branch = br;
				memcpy(tbr + k, br, sizeof(struct branch));
				k++;
			}
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					// insert to stack
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
			}
		}
		else {
			// insert to stack
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			nd = (struct node*)nd->child_node[i];
		}
	}*/
	/*nd = node;
	i = 0;
	for (int j = 0; j <= pos; ++j)
		nd = (struct node*)(nd->child_node[0]);
	for (indexer j = 0; j < count1[pos]; ++j) {
		for (indexer k = 0; k < nd[j].count_child_nodes; ++k) {
			memcpy(tbr + i, nd[j].child_node[k], sizeof(struct branch));
			i++;
		}
	}
	*/
	memcpy(tbr, first_branch, sizeof(struct branch) * m_count_branches);

	// copy data of branches to device
	clock_t t1 = clock();
	hipStream_t stream;
	hipStreamCreate(&stream);
	for (indexer i = 0; i < m_count_branches; ++i)
	{
		void *data_ptr = tbr[i].leaf_x;
		er1 = hipMalloc((void**)&(tbr[i].leaf_x), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_x, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_y;
		er1 = hipMalloc((void**)&(tbr[i].leaf_y), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_y, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_number;
		er1 = hipMalloc((void**)&(tbr[i].leaf_number), sizeof(indexer) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_number, data_ptr, sizeof(indexer) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].merge_next_leaf;
		er1 = hipMalloc((void**)&(tbr[i].merge_next_leaf), sizeof(bool) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].merge_next_leaf, data_ptr, sizeof(bool) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].xsh_min;
		er1 = hipMalloc((void**)&(tbr[i].xsh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].xsh_max;
		er1 = hipMalloc((void**)&(tbr[i].xsh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_min;
		er1 = hipMalloc((void**)&(tbr[i].ysh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_max;
		er1 = hipMalloc((void**)&(tbr[i].ysh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].offset;
		er1 = hipMalloc((void**)&(tbr[i].offset), sizeof(indexer) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].offset, data_ptr, sizeof(indexer) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
	}
	hipStreamSynchronize(stream);
	hipStreamDestroy(stream);
	clock_t t2 = clock();
	printf("Time copying data to device = %u ms\n", t2 - t1);

	// copy branches to device
	struct branch *dev_br = NULL;
	er1 = hipMalloc((void**)&dev_br, sizeof(struct branch) * m_count_branches);
	er1 = hipMemcpy(dev_br, tbr, sizeof(struct branch) * m_count_branches, hipMemcpyHostToDevice);

	//return false;
	alignas(16) struct node *to_dev_nd[65];
	//void **to_dev_child[64];
	struct node *dev_nd = NULL, *dev_nd_prev = NULL, *dev_ptr = NULL;
	// to_dev_nd[0] = (struct node*)aligned_alloc(16, sizeof(struct node));
	// memcpy(to_dev_nd[0], nd, sizeof(struct node));
	struct node* tnd = node;
	//for (unsigned j = 0; j <= pos; ++j)
	//	tnd = (struct node*)(tnd->child_node[0]);
	//unsigned j = 0;
	//void* tmp1 = NULL;
	unsigned count = tnd->count_child_nodes, prev_count = 1;
	for (int k1 = pos; k1 >= 0; --k1) {
		tnd = node;
		//for (unsigned j = 0; j <= k1; ++j)
		//	tnd = (struct node*)(tnd->child_node[0]);
		// data child node
		to_dev_nd[k1] = (struct node*)aligned_alloc(16, sizeof(struct node) * count1[k1]);
		//memcpy(to_dev_nd[k1], tnd/*->child_node[0]*/, sizeof(struct node) * count1[k1]);
		memcpy(to_dev_nd[k1], stack_first_node[k1 + 1], sizeof(struct node) * count1[k1]);
		// pointer to child_node on host
		for (indexer j = 0; j < count1[k1]; ++j) {
			//(to_dev_nd[k1])[j]->child_node = (void**)aligned_alloc(16, sizeof(void*) * MAX_NODES); // tnd->count_child_nodes);
			//(to_dev_child[k1])[j] = to_dev_nd[j]->child_node;
			dev_ptr = NULL;
			er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * MAX_NODES);
			(to_dev_nd[k1])[j].child_node = (void**)dev_ptr;
			for (indexer k2 = 0; k2 < MAX_NODES; ++k2) {
				if (k1 == pos) {
					// copy pointer of branches
					//struct branch *ptr = &(dev_br[k2 + j * MAX_NODES]);
					unsigned idx = (struct branch*)((struct node*)(stack_first_node[k1 + 1])[j].child_node[k2]) - first_branch;
					struct branch *ptr = &(dev_br[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct branch*), hipMemcpyHostToDevice);
				}
				else {
					// copy pointer of nodes
					//struct node* ptr = &(dev_nd_prev[k2 + j * MAX_NODES]);
					unsigned idx = (struct node*)(stack_first_node[k1 + 1])[j].child_node[k2] - (struct node*)(stack_first_node[k1 + 2]);
					//printf("%u\n", idx);
					struct node *ptr = &(dev_nd_prev[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
				}
			}
		}
		//printf("==========================================\n\n\n");
		// pointers of child nodes
		er1 = hipMalloc((void**)&dev_nd, sizeof(struct node) * count1[k1]); // tnd->count_child_nodes);
		hipMemcpy(dev_nd, to_dev_nd[k1], sizeof(struct node) * count1[k1], hipMemcpyHostToDevice);
		dev_nd_prev = dev_nd;
	}
	// copy top node (root)
	to_dev_nd[64] = (struct node*)aligned_alloc(16, sizeof(struct node));
	memcpy(to_dev_nd[64], node/*->child_node[0]*/, sizeof(struct node));
	dev_ptr = NULL;
	er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * node->count_child_nodes);
	(to_dev_nd[64])[0].child_node = (void**)dev_ptr;
	for (indexer k2 = 0; k2 < node->count_child_nodes; ++k2) {
		// copy pointer of nodes
		//struct node* ptr = &(dev_nd_prev[k2]);
		unsigned idx = (struct node*)(stack_first_node[0])[0].child_node[k2] - (struct node*)(stack_first_node[1]);
		struct node* ptr = &(dev_nd_prev[idx]);
		er1 = hipMemcpy((void*)((to_dev_nd[64])[0].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
	}
	// pointers of child nodes
	er1 = hipMalloc((void**)&dev_nd, sizeof(struct node)); // tnd->count_child_nodes);
	er1 = hipMemcpy(dev_nd, to_dev_nd[64], sizeof(struct node), hipMemcpyHostToDevice);
	m_dev_node = dev_nd;
	printf("============== 0x%llx, 0x%llx, prev = 0x%llx\n", m_dev_node, dev_nd, dev_nd_prev);

	// free memory
	for (int k1 = pos; k1 >= 0; --k1) {
		_aligned_free(to_dev_nd[k1]);
	}
	_aligned_free(to_dev_nd[64]);

	// allocating memory for root
	//er1 = hipMalloc((void**)&m_dev_node, sizeof(struct node));
	// copy to device root of tree
	//hipMemcpy(m_dev_node, to_dev_nd[0], sizeof(struct node), hipMemcpyHostToDevice);

	return true;
}

extern "C"
bool destroy_cuda_device()
{
	hipFree(dev_threads_count);
	hipError_t er1 = hipDeviceReset();
	return er1 == hipSuccess ? true : false;
}

extern "C"
#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
indexer* cuda_search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items, ret_callback2_circle callback = NULL, void *data = NULL);
/* searchin items in selected rectangle on cuda device imlementation */
__global__ void cuda_search_rect2_impl1(void **nd, indexer *iter_count, indexer *atomic_iter, /*out*/ void **next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items, ret_callback2_circle callback = NULL, void *data = NULL);
__global__ void cuda_search_rect2_impl2(void **nd, indexer *iter_count, indexer *atomic_iter, /*out*/ void **next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items, ret_callback2_circle callback = NULL, void *data = NULL);
#else
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
__global__ indexer* search_rect2_impl(void *nd_ptr, indexer iter_count, /*out*/indexer *count_items)
#endif // CALC_POINT

#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
indexer* cuda_search_rect2(node * nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, indexer * count_items, ret_callback2_circle callback, void * data)
#else
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
#endif // CALC_POINT
{

	// memory for result
	/*size_t mem_size = 256;
	size_t count_mem = 1;
	alignas(16) indexer* idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * mem_size * count_mem); */
	hipError_t er1;
	indexer *host_idxs = NULL, *dev_idxs = NULL;
	hipHostAlloc((void**)&host_idxs, sizeof(indexer) * MAX_RESULTS, hipHostMallocMapped);
	hipHostGetDevicePointer((void**)&dev_idxs, host_idxs, 0);

	// searching
	hipEvent_t start, stop;
	float gtime = 0.0;
	int device_id;
	hipDeviceProp_t prop;
	er1 = hipGetDevice(&device_id);
	er1 = hipGetDeviceProperties(&prop, device_id);
	dim3 grid_size = dim3(prop.multiProcessorCount, 1, 1), block_size = dim3(prop.warpSize, 1, 1);
	// store boundaries
	boundaries b1;
	b1.intersection = intersection; b1.x_max = x_max; b1.x_min = x_min; b1.y_max = y_max; b1.y_min = y_min;
	//hipMalloc((void**)dev_bonds, sizeof(struct boundaries));
	hipMemcpyToSymbol(HIP_SYMBOL(dev_bonds), &b1, sizeof(struct boundaries));
	// for store count of iterations to next step
	indexer *dev_atomic_iter = NULL;
	hipMalloc((void**)&dev_atomic_iter, sizeof(indexer));
	hipMemset(dev_atomic_iter, 0, 1);
	// store pointers for next step
	void **dev_ptr = NULL, **dev_ptr2 = NULL;
	hipMalloc((void**)&dev_ptr, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx, count_br = %u\n", &m_dev_node, m_dev_node, m_count_branches);
	void **tptr = (void**)(&m_dev_node);
	hipMemcpy(dev_ptr, tptr, sizeof(void*), hipMemcpyHostToDevice);
	hipMalloc((void**)&dev_ptr2, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx; dev_ptr = 0x%llx\n", &m_dev_node, m_dev_node, dev_ptr);
	// count items
	indexer *dev_count_items = NULL;
	hipMalloc((void**)&dev_count_items, sizeof(indexer));
	// count of iterations
	indexer *dev_iter_count = NULL;
	hipMalloc((void**)&dev_iter_count, sizeof(indexer));
	hipMemset(dev_iter_count, 1, 1);

	hipStream_t stream;
	hipStreamCreate(&stream);
	indexer atomic_iter = 0;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, stream);
	
	// calculating nodes
	for (int i = 0; i < m_length_of_tree + 1; ++i) {
		er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
		cuda_search_rect2_impl1 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_iter_count, dev_atomic_iter, dev_ptr2, host_idxs, count_items, callback, data);

		er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
		er1 = hipMemcpyAsync(dev_ptr, dev_ptr2, sizeof(void*) * atomic_iter, hipMemcpyDeviceToDevice, stream);
		er1 = hipMemcpyAsync(dev_iter_count, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToDevice, stream);
		printf("Iter %i: next = %u (%s)\n", i, atomic_iter, er1 == hipSuccess ? "true" : "false");
		hipStreamSynchronize(stream);
	}

	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 1 time = %f ms\n", gtime);

	// calculating branches
	grid_size = dim3(atomic_iter, 1, 1);
	/*int device_id;
	hipDeviceProp_t prop;
	er1 = hipGetDevice(&device_id);
	er1 = hipGetDeviceProperties(&prop, device_id);
	if (MAX_NODES <= prop.maxThreadsPerBlock)
		block_size = dim3(MAX_NODES, 1, 1);
	else
		block_size = dim3(prop.maxThreadsPerBlock, 1, 1);*/

	hipEventRecord(start, stream);
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	cuda_search_rect2_impl2 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_iter_count, dev_atomic_iter, dev_ptr2, host_idxs, count_items, callback, data);
	er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
	hipStreamSynchronize(stream);
	hipDeviceSynchronize();
	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 2 time = %f ms\n", gtime);

	hipMemcpyAsync(count_items, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);

	indexer *idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * *count_items);
	hipMemcpyAsync(idxs, host_idxs, sizeof(indexer) * *count_items, hipMemcpyHostToHost, stream);
	hipStreamSynchronize(stream);

	// freeing and destroying
	hipStreamDestroy(stream);

	hipFree(dev_count_items);
	hipFree(dev_ptr);
	hipFree(dev_atomic_iter);
	hipEventDestroy(stop);
	hipEventDestroy(start);
	hipHostFree(host_idxs);
	return idxs;
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl1(void **nd_ptr, indexer *iter_count, indexer *atomic_iter, /*out*/ void** next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items, ret_callback2_circle callback, void *data)
{
	int idxx = threadIdx.x + blockIdx.x * blockDim.x;
	//printf("Thread = %u\n", idxx);
	//int idxx_t = idxx % (*dev_threads_count);
	indexer iter_count_t = *iter_count % (*dev_threads_count);

	//if (!idxx)
	//	printf("Calc NODE iter_count = %u (0x%llx)\n", *iter_count, iter_count);

	struct node* nd = (struct node*)nd_ptr[0];
	//indexer idx = 0;

	//struct node *stack_node[64];
	//int stack_pos = 0;
	//indexer stack_idx[64];

	//coord tx = 0.0, ty = 0.0, tx1 = 0.0, ty1 = 0.0;
	//indexer tn = 0, tn1 = 0;

	//indexer i = 0;

	//coord dist = 0.0;

#ifdef CALC_POINT
	//coord tmp_dist = FLT_MAX;
	//indexer tmp_idx = -1;
#endif // CALC_POINT
	int t = (int)ceilf((float)*iter_count / (float)*dev_threads_count);
	int t1 = t - 1;
	for (int j = 0; j < t; ++j) {
		//printf("Thread = %i, j = %u\n", idxx, j);
		if (j == t1 && idxx >= iter_count_t) {// idxx_t >= *iter_count) {
			return;
		}
		//printf("Thread %i (%i): x1 = %f, y1 = %f, x2 = %f, y2 = %f\n", idxx + j * (*dev_threads_count), j, nd[idxx + j * (*dev_threads_count)].x1, nd[idxx + j * (*dev_threads_count)].y1, nd[idxx + j * (*dev_threads_count)].x2, nd[idxx + j * (*dev_threads_count)].y2);
		// node in bounrary or bounrary in node
		if (nd[idxx + j * (*dev_threads_count)].x1 <= dev_bonds->x_max && nd[idxx + j * (*dev_threads_count)].x2 >= dev_bonds->x_min && nd[idxx + j * (*dev_threads_count)].y1 <= dev_bonds->y_max && nd[idxx + j * (*dev_threads_count)].y2 >= dev_bonds->y_min) {
			//printf("Thread %i (%i) ================================ ====\n", idxx + j * (*dev_threads_count), j);
			// check node fully in the boundary
			//if (nd[idxx].x1 >= dev_bonds->x_min && nd[idxx].y1 >= dev_bonds->y_min && nd[idxx].x2 <= dev_bonds->x_max && nd[idxx].y2 <= dev_bonds->y_max) {
				// node is fully in the boundary
			//}
			//else {
				// node isn't fully in the boundary, than add to calculation to next iteration
				indexer t1 = atomicAdd(atomic_iter, nd[idxx + j * (*dev_threads_count)].count_child_nodes);
				//printf("Increase %i: %u to %u (%u)\n", idxx, t1, *atomic_iter, nd[idxx].count_child_nodes);
				/*if (t1 + nd->count_child_nodes >= 10000)
					return; */
				for (unsigned k = t1, t2 = 0; k < t1 + nd[idxx + j * (*dev_threads_count)].count_child_nodes; ++k, ++t2) {
					//void **ptr = &next_nd;
					next_nd[k] = nd[idxx + j * (*dev_threads_count)].child_node[t2];
				}
			//}
		}
		else {
			// node and boundary isn't intersection
		}
	}

//	return 1;
/*	while (i < nd->count_child_nodes) {
		// node in bounrary or bounrary in node
		if (nd->x1 <= x_max && nd->x2 >= x_min && nd->y1 <= y_max && nd->y2 >= y_min) {
			// check node fully in the boundary
#if defined(CALC_CIRCLE) || defined(CALC_POINT)
			if (!callback2 && nd->x1 >= x_min && nd->y1 >= y_min && nd->x2 <= x_max && nd->y2 <= y_max) {
#else
			if (nd->x1 >= x_min && nd->y1 >= y_min && nd->x2 <= x_max && nd->y2 <= y_max) {
#endif // CALC_CIRCLE
				// node is fully in the boundary
				add_nodes(nd, &mem_size, &count_mem, &idx, &idxs);
#ifdef MINIMAL_DEBUG2
				printf("Increment node on %u\n", idx - cc3);
				cc3 = idx;
#endif // MINIMAL_DEBUG2
			}
			else {
				// node not fully in the boundaty
				if (nd->is_last_node) {
					for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
						struct branch *br = (struct branch*)(nd->child_node)[j];
						// checking like node
						if (br->x_min <= x_max && br->x_max >= x_min && br->y_min <= y_max && br->y_max >= y_min) {
							// check branch fully in the boundary
#if defined(CALC_CIRCLE) || defined(CALC_POINT)
							if (!callback2 && br->x_min >= x_min && br->y_min >= y_min && br->x_max <= x_max && br->y_max <= y_max) {
#else
							if (br->x_min >= x_min && br->y_min >= y_min && br->x_max <= x_max && br->y_max <= y_max) {
#endif // CALC_CIRCLE
								// branch is fully in the boundary
								add_branch(br, &mem_size, &count_mem, &idx, &idxs);
#ifdef MINIMAL_DEBUG2
								printf("Increment branch on %u\n", idx - cc3);
								cc3 = idx;
#endif // MINIMAL_DEBUG2
							}
							else {
								for (indexer i1 = 0; i1 < br->count_shapes; ++i1) {
									// checking like node
									if (br->xsh_min[i1] <= x_max && br->xsh_max[i1] >= x_min && br->ysh_min[i1] <= y_max && br->ysh_max[i1] >= y_min) {
										// check shape fully in boundary or intersection with boundary
										bool fl1 = false;
										indexer end = i1 + 1 >= br->count_shapes ? br->count_leafs : br->offset[i1 + 1];
										for (indexer k = br->offset[i1]; k < end; ++k) {
											if (br->leaf_x[k] >= x_min && br->leaf_x[k] <= x_max && br->leaf_y[k] >= y_min && br->leaf_y[k] <= y_max) {
												fl1 = true;
												break;
											}
										}
										if (!fl1 && intersection) {
											// last check: intersection

											// side 1/2
											for (indexer k = br->offset[i1]; k < end; ++k) {
												if (k != end - 1) {
													if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[k + 1], br->leaf_y[k + 1], x_min, y_min, x_max, y_min)) {
														fl1 = true;
														break;
													}
												}
												else {
													if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[br->offset[i1]], br->leaf_y[br->offset[i1]], x_min, y_min, x_max, y_min)) {
														fl1 = true;
														break;
													}
												}
											}

											// side 2/3
											if (!fl1) {
												for (indexer k = br->offset[i1]; k < end; ++k) {
													if (k != end - 1) {
														if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[k + 1], br->leaf_y[k + 1], x_max, y_min, x_max, y_max)) {
															fl1 = true;
															break;
														}
													}
													else {
														if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[br->offset[i1]], br->leaf_y[br->offset[i1]], x_max, y_min, x_max, y_max)) {
															fl1 = true;
															break;
														}
													}
												}
											}

											// side 3/4
											if (!fl1) {
												for (indexer k = br->offset[i1]; k < end; ++k) {
													if (k != end - 1) {
														if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[k + 1], br->leaf_y[k + 1], x_max, y_max, x_min, y_max)) {
															fl1 = true;
															break;
														}
													}
													else {
														if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[br->offset[i1]], br->leaf_y[br->offset[i1]], x_max, y_max, x_min, y_max)) {
															fl1 = true;
															break;
														}
													}
												}
											}

											// side 4/1
											if (!fl1) {
												for (indexer k = br->offset[i1]; k < end; ++k) {
													if (k != end - 1) {
														if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[k + 1], br->leaf_y[k + 1], x_min, y_max, x_min, y_min)) {
															fl1 = true;
															break;
														}
													}
													else {
														if (check_intersection(br->leaf_x[k], br->leaf_y[k], br->leaf_x[br->offset[i1]], br->leaf_y[br->offset[i1]], x_min, y_max, x_min, y_min)) {
															fl1 = true;
															break;
														}
													}
												}
											}
										}

#ifdef MINIMAL_DEBUG2
										if (fl1)
											cc4++;
#endif

#ifdef CALC_CIRCLE
										// check callback function to store in return collection
										if (fl1 && callback2 && data) {
											struct calc_data cc;
											cc.br = br;
											cc.idx = i1;
											struct point *center = (struct point*)data;
											cc.center = struct point(*center);
											cc.radius = (x_max - x_min) / 2.0;

											fl1 = callback2(&cc);
#ifdef MINIMAL_DEBUG2
											cc1++;
											if (fl1)
												cc2++;
#endif // MINIMAL_DEBUG2
										}
#elif defined(CALC_POINT)
										// check callback function to store in return collection
										if (fl1 && callback2 && data) {
											struct calc_data cc;
											cc.br = br;
											cc.idx = i1;
											struct point *center = (struct point*)data;
											cc.center = point(*center);
											//cc.radius = (x_max - x_min) / 2.0;
											cc.dist = FLT_MAX;
											cc.curr_idx = -1;

											fl1 = callback2(&cc);
											if (cc.dist < tmp_dist) {
												tmp_dist = cc.dist;
												tmp_idx = cc.curr_idx;
											}
											idx = 1;
										}

#endif //CALC_CIRCLE

										if (fl1) {
#ifdef MINIMAL_DEBUG2
											cc3++;
#endif // MINIMAL_DEBUG2
											// check memory and store index
											if (idx > mem_size * count_mem) {
												count_mem++;
												idxs = (indexer*)_aligned_realloc(idxs, sizeof(indexer) * mem_size * count_mem, 16);
											}
											// store index of current sergment from shape
											idxs[idx] = br->leaf_number[br->offset[i1]];
											idx++;

										}
									}
								}
							}
							}
						}
					// return from stack
					while (stack_pos > 0) {
						stack_pos--;
						nd = stack_node[stack_pos];
						i = stack_idx[stack_pos] + 1;

						if (i < nd->count_child_nodes) {
							stack_idx[stack_pos] = i;
							stack_node[stack_pos] = nd;
							stack_pos++;
							nd = (struct node*)nd->child_node[i];
							i = 0;
							break;
						}
					}
					}
				else {
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					i = 0;
					nd = (struct node*)nd->child_node[i];
				}
				}
			}
		else if (stack_pos > 0) {
			// return from stack
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
			}
		}
		else {
			char ch1[128];
#ifndef _WIN
			snprintf(ch1, 128, "node: x_min=%.2f, y_min=%.2f, x_max=%.2f, y_max=%.2f, rect: x_min=%.2f, y_min=%.2f, x_max=%.2f, y_max=%.2f", nd->x1, nd->y1, nd->x2, nd->x2, x_min, y_min, x_max, y_max);
#else
			sprintf_s(ch1, 128, "node: x_min=%.2f, y_min=%.2f, x_max=%.2f, y_max=%.2f, rect: x_min=%.2f, y_min=%.2f, x_max=%.2f, y_max=%.2f", nd->x1, nd->y1, nd->x2, nd->x2, x_min, y_min, x_max, y_max);
#endif
			lprintf(ch1);
			return NULL; //(indexer)-1;
		}
		//continue;
			};

	//free(stack_node);
	//free(stack_idx);
#ifdef MINIMAL_DEBUG
	char ch1[128];
	//sprintf_s(ch1, 128, "x=%.2f, y=%.2f, point: x=%.2f, y=%.2f, x=%.2f, y=%.2f dist=%.2f, n1=%u, n2=%u", x, y, tx, ty, tx1, ty1, tres.dist, tn, tn1);
	sprintf_s(ch1, 128, "c1 = %u, c2 = %u, utilization = %0.2f", temp_counter1, temp_counter2, temp_counter2 * 100.0 / temp_counter1);
	lprintf(ch1);
#endif
	*count_items = idx;
	idxs = (indexer*)_aligned_realloc(idxs, sizeof(indexer) * idx, 16);
#ifdef MINIMAL_DEBUG2
	printf("DEBUG: count1 = %u, count2 = %u, count3 = %u, count4 = %u\n", cc1, cc2, cc3, cc4);
#endif

#ifdef CALC_POINT
	idxs[0] = tmp_idx;
	if (data) {
		struct point *center = (struct point*)data;
		center->x = tmp_dist;
	}
#endif
	return idxs; //(indexer)-1;
	*/
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl2(void **br_ptr, indexer *iter_count, indexer *atomic_iter, /*out*/ void** next_nd, /*out*/ indexer *idxs, /*out*/indexer *count_items, ret_callback2_circle callback, void *data)
{
	int idxx = threadIdx.x;
	int idx_gr_br = blockIdx.x;
	//printf("Thread = %u\n", idxx);
	//if (!idxx /*&& !idx_gr_br*/)
	//	printf("============================================== block = %d, %u\n", idx_gr_br, *atomic_iter);

	// for store temporary results
	__shared__ indexer temp_res[33]; // must be as blockDim.x size + 1 (for rpevious result)
	__shared__ char temp_res_flag[32];
	temp_res[idxx] = (indexer)-1;
	if (!idxx)
		temp_res[32] = (indexer)-1;
	temp_res_flag[idxx] = -1;
	//__shared__ int index = 0;

	//if (!idxx)
	//	printf("Calc BR iter_count = %u (0x%llx), grid.x = %i\n", *iter_count, iter_count, idx_gr_br);

	struct branch* br = (struct branch*)br_ptr[0];

	if (br[idx_gr_br].x_min <= dev_bonds->x_max && br[idx_gr_br].x_max >= dev_bonds->x_min && br[idx_gr_br].y_min <= dev_bonds->y_max && br[idx_gr_br].y_max >= dev_bonds->y_min) {
		//printf("%i, %i\n", idx_gr_br, idxx);
		//int idxx_t = idxx % blockDim.x;
		int t = (int)ceilf((float)br[idx_gr_br].count_leafs / (float)blockDim.x);
		int t1 = t - 1;
		for (int j = 0; j < t; ++j) {
			int curr_offset = j * blockDim.x;
			if (j == t1 && idxx + curr_offset >= br[idx_gr_br].count_leafs) {
				return;
			}

			// check points to enter to boundary
			if (br[idx_gr_br].leaf_x[idxx + curr_offset] >= dev_bonds->x_min && br[idx_gr_br].leaf_x[idxx + curr_offset] <= dev_bonds->x_max && br[idx_gr_br].leaf_y[idxx + curr_offset] >= dev_bonds->y_min && br[idx_gr_br].leaf_y[idxx + curr_offset] <= dev_bonds->y_max) {
				temp_res[idxx] = br[idx_gr_br].leaf_number[idxx + curr_offset];
				/*int t2 = atomicAdd(atomic_iter, 1);
				if (t2 >= MAX_RESULTS - 1) {
					// can not store result
					atomicSub(atomic_iter, 1);
				}
				else {
					// can store result
					//idxs[t2] = 
				}*/
			}
			__syncthreads();

			// packing temporary results
			if (temp_res[idxx] == temp_res[idxx + 1]) {
				__threadfence();
				temp_res[idxx + 1] = -1;
				//printf("Index of equals items = %u + %u => %u\n", idxx, idxx + 1, temp_res[idxx + 1]);
			}
			else {
				//__threadfence();
			}
			__syncthreads();
			// ckeck for previous result
			if (!idxx) {
				//if (temp_res[0] != (unsigned)-1) printf("%i:%i Prev step = %u (%u)\n", idx_gr_br, j, temp_res[32], temp_res[0]);
				if (temp_res[32] == temp_res[0]) {
					//if (temp_res[0] != (unsigned)-1) printf("From prev step = %u\n", temp_res[0]);
					temp_res[0] = -1;
				}
			}
			__syncthreads();

			// store temporary results to global array
			if (temp_res[idxx] != -1) {
				int t2 = atomicAdd(atomic_iter, 1);
				if (t2 >= MAX_RESULTS - 1) {
					// can not store result
					atomicSub(atomic_iter, 1);
				}
				else {
					// can store result
					idxs[t2] = temp_res[idxx];
					//printf("Number = %u, value = %u (%u)\n", t2, temp_res[idxx], idxs[t2]);
					temp_res_flag[idxx] = idxx;
				}
			}

			__syncthreads();

			// store previous result
			for (int t2 = blockDim.x / 2; t2 > 0; t2 >>= 1)
			{
				if (idxx < t2) {
					if (temp_res_flag[idxx] < temp_res_flag[idxx + t2])
						temp_res_flag[idxx] = temp_res_flag[idxx + t2];
				}
				__syncthreads();
			}
			if (!idxx) {
				if (temp_res_flag[idxx] != -1) {
					//printf("INDEX === %u\n", temp_res_flag[idxx]);
					temp_res[32] = temp_res[temp_res_flag[idxx]]; // idxs[idxx];
				}
				//if (temp_res[32] != (unsigned)-1) printf("%i:%i Next step = %u\n", idx_gr_br, j, temp_res[32]);
			}
			
			// reset temporary resulats
			temp_res[idxx] = -1;
			temp_res_flag[idxx] = -1;
			__syncthreads();
		}
	}
}