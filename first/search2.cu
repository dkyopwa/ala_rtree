#include "hip/hip_runtime.h"
/*

*/

#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <time.h>
#include <math.h>
#include <float.h>
//#include <hip/hip_runtime.h>
#include "unimem.h"
#include "first.h"

//#define DEBUG_CUDA
#define MAX_RESULTS 100000
#define PACK_RESULTS

struct boundaries {
	coord x_min;
	coord y_min;
	coord x_max;
	coord y_max;
	bool intersection;
};

struct node* m_dev_node = NULL;
__constant__ boundaries dev_bonds[1];
__constant__ unsigned dev_threads_count[1];
unsigned m_threads_count;
indexer m_count_branches;
int m_length_of_tree = 0;

//__constant__ struct branch *m_ttt_cuda_first_branch = NULL;

#ifdef PACK_RESULTS
/// compare
int cmp(const void* a, const void* b)
{
	return (int)(*(indexer*)a - *(indexer*)b);
}

#endif

extern "C"
bool init_cuda_device(int deviceID, struct node* node)
{
	if (!node)
		return false;
	//return false;

	struct node *nd = node;
	unsigned count1[64], i = 0;
	m_count_branches = 0;
	for (int j = 0; j < 64; j++) count1[j] = 0;
	//count1[0] = 1;

	struct node *stack_node[64];
	int stack_pos = 0;
	indexer stack_idx[64];
	alignas(16) struct branch *first_branch = NULL;
	alignas(16) struct node* stack_first_node[64];
	for (unsigned i = 0; i < 64; ++i) {
		stack_first_node[i] = NULL;
	}
	while (i < nd->count_child_nodes) {
		if (!stack_first_node[stack_pos] || nd < stack_first_node[stack_pos])
			stack_first_node[stack_pos] = nd;
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				if (!first_branch || br < first_branch)
					first_branch = br;
			}
			/*if (!count_br)
				count_br = nd->count_child_nodes;
			if (count_br != nd->count_child_nodes)
				printf("Branches %u vs %u\n", count_br, nd->count_child_nodes);*/
			m_count_branches += nd->count_child_nodes;
				// return from stack
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
				else {
					//if (count1[stack_pos]) {
						//if (count1[stack_pos] != nd->count_child_nodes) {
						//	printf("Nodes %u vs %u\n", count1[stack_pos], nd->count_child_nodes);
						//}
						count1[stack_pos] += nd->count_child_nodes;
					//}
					//else {
					//	count1[stack_pos] = nd->count_child_nodes;
					//}
				}
			}
		}
		else {
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			i = 0;
			nd = (struct node*)nd->child_node[i];
			/*if (!count1[stack_pos])
				count1[stack_pos] = nd->count_child_nodes;
			else
				count1[stack_pos] += nd->count_child_nodes;*/
		}
		/*} else if (i < nd->count_child_nodes) {
		i++;*/
		/*if (!count1[stack_pos])
			count1[stack_pos] = nd->count_child_nodes;
		else
			count1[stack_pos] += nd->count_child_nodes;
			*/
	}

	//return false;

	int deviceCount;
	hipError_t er1 = hipGetDeviceCount(&deviceCount);
	printf("DevicecheckCudaErrors Count: %d\n", deviceCount);

	if (deviceID == -1)
		deviceID = 0;

	hipDeviceProp_t prop;
	for (int ii = 0; ii < deviceCount; ++ii) {
		er1 = hipGetDeviceProperties(&prop, ii);
		if (prop.major < 2 || prop.canMapHostMemory != 1)
		{
			printf("ERROR: calculation requires GPU devices with compute SM 2.0 or higher, or can not using MapHostMemory.\n");
			printf("Current GPU device has compute SM%d.%d, Exiting...", prop.major, prop.minor);
			//exit(EXIT_WAIVED);
			return false;
		}

		printf("GPU device name is %s\n", prop.name);
		printf("GPU total memory = %.0f Mb\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		printf("Number of multiprocessors on the device = %u\n", prop.multiProcessorCount);
	}

	er1 = hipSetDevice(deviceID);
	hipSetDeviceFlags(hipDeviceMapHost);
	er1 = hipGetDeviceProperties(&prop, deviceID);
	m_threads_count = prop.multiProcessorCount * prop.warpSize * 2;
	//er1 = hipMalloc((void**)&dev_threads_count, sizeof(unsigned));
	er1 = hipMemcpyToSymbol(HIP_SYMBOL(dev_threads_count), &m_threads_count, sizeof(unsigned));

	// copy rtree
	int pos = 63;

	for (; pos >= 0; --pos) {
		if (count1[pos])
			break;
	}
	m_length_of_tree = pos + 1;

	// allocationg memory for branches
	alignas(16) struct branch* tbr = (struct branch*)aligned_alloc(16, sizeof(struct branch) * m_count_branches);
	// struct branch* first_branch = NULL;
	/*nd = node;
	i = 0;
	unsigned k = 0;
	while (i < nd->count_child_nodes) {
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				//if (!first_branch || br < first_branch)
				//	first_branch = br;
				memcpy(tbr + k, br, sizeof(struct branch));
				k++;
			}
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					// insert to stack
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
			}
		}
		else {
			// insert to stack
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			nd = (struct node*)nd->child_node[i];
		}
	}*/
	/*nd = node;
	i = 0;
	for (int j = 0; j <= pos; ++j)
		nd = (struct node*)(nd->child_node[0]);
	for (indexer j = 0; j < count1[pos]; ++j) {
		for (indexer k = 0; k < nd[j].count_child_nodes; ++k) {
			memcpy(tbr + i, nd[j].child_node[k], sizeof(struct branch));
			i++;
		}
	}
	*/
	memcpy(tbr, first_branch, sizeof(struct branch) * m_count_branches);

	// for debug
	/*printf("\n\n\n======================================================================\n");
	for (indexer i = 0; i < count1[pos]; ++i) {
		if ((struct node*)(stack_first_node[pos + 1])[i].is_last_node) {
			unsigned tt = ((struct node*)(stack_first_node[pos + 1]))[i].count_child_nodes;
			for (indexer ii = 0; ii < tt; ++ii) {
				unsigned idx = (struct branch*)((struct node*)(stack_first_node[pos + 1])[i].child_node[ii]) - first_branch;
				if (!idx)
					printf("0\n");
				else
					printf("%u\n", idx);
			}
		}
		else {
			printf("Error last node %u\n", i);
		}
	}*/

	// copy data of branches to device
	clock_t t1 = clock();
	hipStream_t stream;
	hipStreamCreate(&stream);
	for (indexer i = 0; i < m_count_branches; ++i)
	{
		void *data_ptr = tbr[i].leaf_x;
		er1 = hipMalloc((void**)&(tbr[i].leaf_x), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_x, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_y;
		er1 = hipMalloc((void**)&(tbr[i].leaf_y), sizeof(coord) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_y, data_ptr, sizeof(coord) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].leaf_number;
		er1 = hipMalloc((void**)&(tbr[i].leaf_number), sizeof(indexer) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].leaf_number, data_ptr, sizeof(indexer) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].merge_next_leaf;
		er1 = hipMalloc((void**)&(tbr[i].merge_next_leaf), sizeof(bool) * tbr[i].count_leafs);
		er1 = hipMemcpyAsync(tbr[i].merge_next_leaf, data_ptr, sizeof(bool) * tbr[i].count_leafs, hipMemcpyHostToDevice, stream);
		/*data_ptr = tbr[i].xsh_min;
		er1 = hipMalloc((void**)&(tbr[i].xsh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].xsh_max;
		er1 = hipMalloc((void**)&(tbr[i].xsh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].xsh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_min;
		er1 = hipMalloc((void**)&(tbr[i].ysh_min), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_min, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
		data_ptr = tbr[i].ysh_max;
		er1 = hipMalloc((void**)&(tbr[i].ysh_max), sizeof(coord) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].ysh_max, data_ptr, sizeof(coord) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream); */
		data_ptr = tbr[i].offset;
		er1 = hipMalloc((void**)&(tbr[i].offset), sizeof(indexer) * tbr[i].count_shapes);
		er1 = hipMemcpyAsync(tbr[i].offset, data_ptr, sizeof(indexer) * tbr[i].count_shapes, hipMemcpyHostToDevice, stream);
	}
	er1 = hipStreamSynchronize(stream);
	er1 = hipStreamDestroy(stream);
	clock_t t2 = clock();
	printf("Time copying data to device = %u ms\n", t2 - t1);

	// copy branches to device
	struct branch *dev_br = NULL;
	er1 = hipMalloc((void**)&dev_br, sizeof(struct branch) * m_count_branches);
	er1 = hipMemcpy(dev_br, tbr, sizeof(struct branch) * m_count_branches, hipMemcpyHostToDevice);
	//hipMemcpyToSymbol(HIP_SYMBOL(m_ttt_cuda_first_branch), &dev_br, sizeof(struct branch*));

	//return false;
	alignas(16) struct node *to_dev_nd[65];
	//void **to_dev_child[64];
	struct node *dev_nd = NULL, *dev_nd_prev = NULL, *dev_ptr = NULL;
	// to_dev_nd[0] = (struct node*)aligned_alloc(16, sizeof(struct node));
	// memcpy(to_dev_nd[0], nd, sizeof(struct node));
	struct node* tnd = node;
	//for (unsigned j = 0; j <= pos; ++j)
	//	tnd = (struct node*)(tnd->child_node[0]);
	//unsigned j = 0;
	//void* tmp1 = NULL;
	unsigned count = tnd->count_child_nodes, prev_count = 1;
	//printf("\n\n\n======================================================================\n");
	for (int k1 = pos; k1 >= 0; --k1) {
		tnd = node;
		//for (unsigned j = 0; j <= k1; ++j)
		//	tnd = (struct node*)(tnd->child_node[0]);
		// data child node
		to_dev_nd[k1] = (struct node*)aligned_alloc(16, sizeof(struct node) * count1[k1]);
		//memcpy(to_dev_nd[k1], tnd/*->child_node[0]*/, sizeof(struct node) * count1[k1]);
		memcpy(to_dev_nd[k1], stack_first_node[k1 + 1], sizeof(struct node) * count1[k1]);
		// pointer to child_node on host
		for (indexer j = 0; j < count1[k1]; ++j) {
			//(to_dev_nd[k1])[j]->child_node = (void**)aligned_alloc(16, sizeof(void*) * MAX_NODES); // tnd->count_child_nodes);
			//(to_dev_child[k1])[j] = to_dev_nd[j]->child_node;
			dev_ptr = NULL;
			er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * MAX_NODES);
			(to_dev_nd[k1])[j].child_node = (void**)dev_ptr;
			for (indexer k2 = 0; k2 < MAX_NODES; ++k2) {
				if (k1 == pos) {
					// copy pointer of branches
					//struct branch *ptr = &(dev_br[k2 + j * MAX_NODES]);
					unsigned idx = (struct branch*)((struct node*)(stack_first_node[k1 + 1])[j].child_node[k2]) - first_branch;
					//if (idx == 4899)
					//	printf("%u\n", idx);
					struct branch *ptr = &(dev_br[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct branch*), hipMemcpyHostToDevice);
				}
				else {
					// copy pointer of nodes
					//struct node* ptr = &(dev_nd_prev[k2 + j * MAX_NODES]);
					unsigned idx = (struct node*)(stack_first_node[k1 + 1])[j].child_node[k2] - (struct node*)(stack_first_node[k1 + 2]);
					//printf("%u\n", idx);
					struct node *ptr = &(dev_nd_prev[idx]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
				}
			}
		}
		//printf("==========================================\n\n\n");
		// pointers of child nodes
		er1 = hipMalloc((void**)&dev_nd, sizeof(struct node) * count1[k1]); // tnd->count_child_nodes);
		hipMemcpy(dev_nd, to_dev_nd[k1], sizeof(struct node) * count1[k1], hipMemcpyHostToDevice);
		dev_nd_prev = dev_nd;
	}
	// copy top node (root)
	to_dev_nd[64] = (struct node*)aligned_alloc(16, sizeof(struct node));
	memcpy(to_dev_nd[64], node/*->child_node[0]*/, sizeof(struct node));
	dev_ptr = NULL;
	er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * node->count_child_nodes);
	(to_dev_nd[64])[0].child_node = (void**)dev_ptr;
	for (indexer k2 = 0; k2 < node->count_child_nodes; ++k2) {
		// copy pointer of nodes
		//struct node* ptr = &(dev_nd_prev[k2]);
		unsigned idx = (struct node*)(stack_first_node[0])[0].child_node[k2] - (struct node*)(stack_first_node[1]);
		struct node* ptr = &(dev_nd_prev[idx]);
		er1 = hipMemcpy((void*)((to_dev_nd[64])[0].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
	}
	// pointers of child nodes
	er1 = hipMalloc((void**)&dev_nd, sizeof(struct node)); // tnd->count_child_nodes);
	er1 = hipMemcpy(dev_nd, to_dev_nd[64], sizeof(struct node), hipMemcpyHostToDevice);
	m_dev_node = dev_nd;
	printf("============== 0x%llx, 0x%llx, prev = 0x%llx\n", m_dev_node, dev_nd, dev_nd_prev);

	// free memory
	for (int k1 = pos; k1 >= 0; --k1) {
		_aligned_free(to_dev_nd[k1]);
	}
	_aligned_free(to_dev_nd[64]);

	// allocating memory for root
	//er1 = hipMalloc((void**)&m_dev_node, sizeof(struct node));
	// copy to device root of tree
	//hipMemcpy(m_dev_node, to_dev_nd[0], sizeof(struct node), hipMemcpyHostToDevice);

	return true;
}

extern "C"
bool destroy_cuda_device()
{
	//hipFree(dev_threads_count);
	hipError_t er1 = hipDeviceReset();
	return er1 == hipSuccess ? true : false;
}

extern "C"
#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
indexer* cuda_search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items);
/* searchin items in selected rectangle on cuda device imlementation */
__global__ void cuda_search_rect2_impl1(void **nd, indexer *iter_count, indexer *atomic_iter, /*out*/ void **next_nd);
__global__ void cuda_search_rect2_impl2(void **br_ptr, indexer *atomic_iter, /*out*/ indexer *idxs);
#else
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
__global__ indexer* search_rect2_impl(void *nd_ptr, indexer iter_count, /*out*/indexer *count_items)
#endif // CALC_POINT

#if defined(CALC_CIRCLE) || defined(CALC_POINT)
/* searchin items in selected rectangle on cuda device */
indexer* cuda_search_rect2(node * nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, indexer * count_items)
#else
indexer* search_rect2(struct node *nd, coord x_min, coord y_min, coord x_max, coord y_max, bool intersection, /*out*/indexer *count_items)
#endif // CALC_POINT
{

	// memory for result
	/*size_t mem_size = 256;
	size_t count_mem = 1;
	alignas(16) indexer* idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * mem_size * count_mem); */
	hipError_t er1;
	hipStream_t stream;
	hipStreamCreate(&stream);

	indexer *host_idxs = NULL, *dev_idxs = NULL, *dev_tmp_idxs = NULL;;
	hipHostAlloc((void**)&host_idxs, sizeof(indexer) * MAX_RESULTS, hipHostMallocMapped);
	hipHostGetDevicePointer((void**)&dev_idxs, host_idxs, 0);
	hipMalloc((void**)&dev_tmp_idxs, sizeof(indexer) * MAX_RESULTS);

	// searching
	hipEvent_t start, stop;
	float gtime = 0.0;
	int device_id;
	hipDeviceProp_t prop;
	er1 = hipGetDevice(&device_id);
	er1 = hipGetDeviceProperties(&prop, device_id);
	dim3 grid_size = dim3(prop.multiProcessorCount, 1, 1), block_size = dim3(prop.warpSize * 2, 1, 1);
	// store boundaries
	boundaries b1;
	b1.intersection = intersection; b1.x_max = x_max; b1.x_min = x_min; b1.y_max = y_max; b1.y_min = y_min;
	//hipMalloc((void**)dev_bonds, sizeof(struct boundaries));
	er1 = hipMemcpyToSymbolAsync(HIP_SYMBOL(dev_bonds), &b1, sizeof(struct boundaries), 0, hipMemcpyHostToDevice, stream);
	// for store count of iterations to next step
	indexer *dev_atomic_iter = NULL;
	er1 = hipMalloc((void**)&dev_atomic_iter, sizeof(indexer));
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	// store pointers for next step
	void **dev_ptr = NULL, **dev_ptr2 = NULL;
	er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx, count_br = %u\n", &m_dev_node, m_dev_node, m_count_branches);
	void **tptr = (void**)(&m_dev_node);
	er1 = hipMemcpyAsync(dev_ptr, tptr, sizeof(void*), hipMemcpyHostToDevice, stream);
	er1 = hipMalloc((void**)&dev_ptr2, sizeof(void*) * m_count_branches);
	//printf("======================= 0x%llx; 0x%llx; dev_ptr = 0x%llx\n", &m_dev_node, m_dev_node, dev_ptr);
	// count items
	indexer *dev_count_items = NULL;
	er1 = hipMalloc((void**)&dev_count_items, sizeof(indexer));
	// count of iterations
	indexer *dev_iter_count = NULL;
	er1 = hipMalloc((void**)&dev_iter_count, sizeof(indexer));
	er1 = hipMemsetAsync(dev_iter_count, 0, sizeof(indexer), stream);
	er1 = hipMemsetAsync(dev_iter_count, 1, 1, stream);
	er1 = hipStreamSynchronize(stream);

	indexer atomic_iter = 1;

#ifdef DEBUG_CUDA
	er1 = hipEventCreate(&start);
	er1 = hipEventCreate(&stop);
	er1 = hipEventRecord(start, stream);
#endif
	
	// calculating nodes
	for (int i = 0; i < m_length_of_tree + 1; ++i) {
		er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
		if (atomic_iter > prop.warpSize * 2) {
			unsigned t = (unsigned)ceil((double)atomic_iter / (double)(prop.warpSize * 2.0));
			block_size = dim3(prop.warpSize * 2, 1, 1);
			grid_size = dim3(t, 1, 1);
		}
		else {
			grid_size = dim3(1, 1, 1);
			block_size = dim3(atomic_iter, 1, 1);
		}
		cuda_search_rect2_impl1 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_iter_count, dev_atomic_iter, dev_ptr2);

		er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
		er1 = hipMemcpyAsync(dev_ptr, dev_ptr2, sizeof(void*) * atomic_iter, hipMemcpyDeviceToDevice, stream);
		er1 = hipMemcpyAsync(dev_iter_count, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToDevice, stream);
		hipStreamSynchronize(stream);
		//printf("===== Iter %i: next = %u (%s)\n", i, atomic_iter, er1 == hipSuccess ? "true" : "false");
		//hipDeviceSynchronize();
	}
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(stop, stream);
	er1 = hipEventSynchronize(stop);
	er1 = hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 1 time = %f ms\n", gtime);
#endif

	// calculating branches
	grid_size = dim3(atomic_iter, 1, 1);
	block_size = dim3(prop.warpSize * 2, 1, 1);
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(start, stream);
#endif
	er1 = hipMemsetAsync(dev_atomic_iter, 0, sizeof(indexer), stream);
	cuda_search_rect2_impl2 << <grid_size, block_size, 0, stream >> > ((void**)dev_ptr, dev_atomic_iter, dev_idxs);
	er1 = hipMemcpyAsync(&atomic_iter, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);
	er1 = hipStreamSynchronize(stream);
	er1 = hipDeviceSynchronize();
#ifdef DEBUG_CUDA
	er1 = hipEventRecord(stop, stream);
	er1 = hipEventSynchronize(stop);
	er1 = hipEventElapsedTime(&gtime, start, stop);
	printf("Kernel 2 time = %f ms\n", gtime);
#endif

	er1 = hipMemcpyAsync(count_items, dev_atomic_iter, sizeof(indexer), hipMemcpyDeviceToHost, stream);

	indexer *idxs = (indexer*)aligned_alloc(16, sizeof(indexer) * *count_items);
	er1 = hipMemcpyAsync(idxs, host_idxs, sizeof(indexer) * *count_items, hipMemcpyHostToHost, stream);
	er1 = hipStreamSynchronize(stream);

	// freeing and destroying
	hipStreamDestroy(stream);

	er1 = hipFree(dev_iter_count);
	er1 = hipFree(dev_ptr);
	er1 = hipFree(dev_ptr2);
	er1 = hipFree(dev_tmp_idxs);
	er1 = hipFree(dev_count_items);
	er1 = hipFree(dev_atomic_iter);
	er1 = hipHostFree(host_idxs);
#ifdef DEBUG_CUDA
	er1 = hipEventDestroy(stop);
	er1 = hipEventDestroy(start);
#endif

#ifdef PACK_RESULTS
	if (*count_items) {
		qsort(idxs, *count_items, sizeof(indexer), cmp);
		indexer j = 1;
		indexer offset = 0;
		for (indexer i = 0; i < *count_items - 1 - offset; ++i) {
			if (idxs[i] == idxs[i + 1 + offset]) {
				offset++;
				idxs[i + 1] = idxs[i + 1 + offset];
				i--;
				continue;
			}
			if (offset)
				idxs[i + 1] = idxs[i + 1 + offset];
			j++;
		}
		*count_items = j;
		idxs = (indexer*)_aligned_realloc(idxs, sizeof(indexer) * j, 16);
	}
#endif
	return idxs;
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl1(void **nd_ptr, indexer *iter_count, indexer *atomic_iter, /*out*/ void** next_nd)
{
	int idxx = threadIdx.x;

	// to temporary store node index
	__shared__ indexer store[64];
	store[threadIdx.x] = (indexer)-1;
	__shared__ int store_idx[1];
	if (!threadIdx.x)
		store_idx[threadIdx.x] = 0;

	struct node** nd = (struct node**)nd_ptr;
	//indexer idx = 0;

#ifdef CALC_POINT
	//coord tmp_dist = FLT_MAX;
	//indexer tmp_idx = -1;
#endif // CALC_POINT
	indexer curr_indexer = idxx + blockIdx.x * blockDim.x; // (*dev_threads_count);
	if (curr_indexer < *iter_count) {
		struct node *curr_nd = nd[curr_indexer];
		__shared__ coord nd_x1[64], nd_x2[64], nd_y1[64], nd_y2[64];
		nd_x1[threadIdx.x] = curr_nd->x1;
		nd_x2[threadIdx.x] = curr_nd->x2;
		nd_y1[threadIdx.x] = curr_nd->y1;
		nd_y2[threadIdx.x] = curr_nd->y2;
			
		// node in bounrary or bounrary in node
		if (nd_x1[threadIdx.x] <= dev_bonds->x_max && nd_x2[threadIdx.x] >= dev_bonds->x_min && nd_y1[threadIdx.x] <= dev_bonds->y_max && nd_y2[threadIdx.x] >= dev_bonds->y_min) {
				// node isn't fully in the boundary, than add to calculation to next iteration
				indexer t3 = atomicAdd(atomic_iter, curr_nd->count_child_nodes);
				//printf("Increase %i: %u to %u (%u)\n", idxx, t1, *atomic_iter, nd[idxx]->count_child_nodes);
				for (unsigned k = t3, t2 = 0; k < t3 + curr_nd->count_child_nodes; ++k, ++t2) {
					next_nd[k] = curr_nd->child_node[t2];
					//printf("Next index = %u\n", (struct branch*)(nd[curr_indexer]->child_node[t2]) - m_ttt_cuda_first_branch);
				}
		}
		else {
			// node and boundary isn't intersection
		}
	}
}

/* searchin items in selected rectangle on cuda device imlementation (step 1) */
__global__ void cuda_search_rect2_impl2(void **br_ptr, indexer *atomic_iter, /*out*/ indexer *idxs)
{
	int idxx = threadIdx.x;
	int idx_gr_br = blockIdx.x;

	// for store temporary results
	__shared__ indexer temp_res[65]; // must be as blockDim.x size + 1 (for rpevious result)
	__shared__ char temp_res_flag[64];
	temp_res[idxx] = (indexer)-1;
	temp_res_flag[idxx] = -1;
	if (!idxx)
		temp_res[64] = (indexer)-1;

	struct branch** br = (struct branch**)br_ptr;
	struct branch *curr_br = br[idx_gr_br];

	if (curr_br->x_min <= dev_bonds->x_max && curr_br->x_max >= dev_bonds->x_min && curr_br->y_min <= dev_bonds->y_max && curr_br->y_max >= dev_bonds->y_min) {
		int t = (int)ceilf((float)curr_br->count_leafs / (float)blockDim.x);
		for (int j = 0; j < t; ++j) {
			int curr_idx = idxx + j * blockDim.x; // curr_offset;
			if (/*j == t1 && */curr_idx < curr_br->count_leafs) {
				// check points to enter to boundary
				if (curr_br->leaf_x[curr_idx] >= dev_bonds->x_min && curr_br->leaf_x[curr_idx] <= dev_bonds->x_max && curr_br->leaf_y[curr_idx] >= dev_bonds->y_min && curr_br->leaf_y[curr_idx] <= dev_bonds->y_max) {
					temp_res[idxx] = curr_br->leaf_number[curr_idx];
				}
				__syncthreads();

				// packing temporary results
				if (temp_res[idxx] == temp_res[idxx + 1]) {
					__threadfence();
					temp_res[idxx + 1] = -1;
				}
				else {
					//__threadfence();
				}
				__syncthreads();

				if (!idxx) {
					if (temp_res[64] == temp_res[0]) {
						temp_res[0] = -1;
					}
				}
				//__syncthreads();

				// store temporary results to global array2
				if (temp_res[idxx] != -1) {
					int t2 = atomicAdd(atomic_iter, 1);
					if (t2 >= MAX_RESULTS - 1) {
						// can not store result
						atomicSub(atomic_iter, 1);
					}
					else {
						// can store result (idxs2 - temporary)
						idxs[t2] = temp_res[idxx];
						temp_res_flag[idxx] = idxx;
					}
				}

				__syncthreads();

				// store previous result
				for (int t2 = blockDim.x / 2; t2 > 0; t2 >>= 1)
				{
					if (idxx < t2) {
						if (temp_res_flag[idxx] < temp_res_flag[idxx + t2])
							temp_res_flag[idxx] = temp_res_flag[idxx + t2];
					}
					__syncthreads();
				}
				if (!idxx) {
					if (temp_res_flag[idxx] != -1) {
						temp_res[64] = temp_res[temp_res_flag[idxx]]; // idxs[idxx];
					}
				}

				// reset temporary resulats
				temp_res[idxx] = -1;
				//temp_res2[idxx] = -1;
				temp_res_flag[idxx] = -1;
				__syncthreads();
			}
		}
	}
}
