/*

*/

#include <stdio.h>
#include <hip/hip_runtime_api.h>
//#include <hip/hip_runtime.h>
#include "unimem.h"
#include "first.h"

#define DEBUG_CUDA_INFO

struct node* m_dev_node = NULL;

extern "C"
bool init_cuda_device(int deviceID, struct node* nd)
{
	if (!nd)
		return false;
	//return false;

	//struct node *tnd = nd;
	unsigned count1[64], i = 0, count_br = 0;
	for (int j = 0; j < 64; j++) count1[j] = 0;
	//count1[0] = 1;

	struct node *stack_node[64];
	int stack_pos = 0;
	indexer stack_idx[64];
	while (i < nd->count_child_nodes) {
		// node in bounrary or bounrary in node
				// node not fully in the boundaty
		if (nd->is_last_node) {
			/*for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node)[j];
			}*/
			/*if (!count_br)
				count_br = nd->count_child_nodes;
			if (count_br != nd->count_child_nodes)
				printf("Branches %u vs %u\n", count_br, nd->count_child_nodes);*/
			count_br += nd->count_child_nodes;
				// return from stack
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
				else {
					//if (count1[stack_pos]) {
						//if (count1[stack_pos] != nd->count_child_nodes) {
						//	printf("Nodes %u vs %u\n", count1[stack_pos], nd->count_child_nodes);
						//}
						count1[stack_pos] += nd->count_child_nodes;
					//}
					//else {
					//	count1[stack_pos] = nd->count_child_nodes;
					//}
				}
			}
		}
		else {
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			i = 0;
			nd = (struct node*)nd->child_node[i];
			/*if (!count1[stack_pos])
				count1[stack_pos] = nd->count_child_nodes;
			else
				count1[stack_pos] += nd->count_child_nodes;*/
		}
		/*} else if (i < nd->count_child_nodes) {
		i++;*/
		/*if (!count1[stack_pos])
			count1[stack_pos] = nd->count_child_nodes;
		else
			count1[stack_pos] += nd->count_child_nodes;
			*/
	}

	//return false;

	int deviceCount;
	hipError_t er1 = hipGetDeviceCount(&deviceCount);
	printf("DevicecheckCudaErrors Count: %d\n", deviceCount);

	if (deviceID == -1)
		deviceID = 0;

	hipDeviceProp_t prop;
	for (int i = 0; i < deviceCount; ++i) {
		er1 = hipGetDeviceProperties(&prop, i);
		if (prop.major < 2)
		{
			printf("ERROR: calculation requires GPU devices with compute SM 2.0 or higher.\n");
			printf("Current GPU device has compute SM%d.%d, Exiting...", prop.major, prop.minor);
			//exit(EXIT_WAIVED);
			return false;
		}

		printf("GPU device name is %s\n", prop.name);
		printf("GPU total memory = %.0f Mb\n", prop.totalGlobalMem / 1024.0 / 1024.0);
		printf("Number of multiprocessors on the device = %u\n", prop.multiProcessorCount);
	}

	er1 = hipSetDevice(deviceID);

	// copy rtree
	int pos = 63;

	for (; pos >= 0; --pos) {
		if (count1[pos])
			break;
	}

	// allocationg memory for branches
	alignas(16) struct branch* tbr = (struct branch*)aligned_alloc(16, sizeof(struct branch) * count_br);
	// struct branch* first_branch = NULL;
	i = 0;
	unsigned k = 0;
	while (i < nd->count_child_nodes) {
		if (nd->is_last_node) {
			for (unsigned j = 0; j < nd->count_child_nodes; ++j) {
				struct branch *br = (struct branch*)(nd->child_node[j]);
				//if (!first_branch || br < first_branch)
				//	first_branch = br;
				memcpy(tbr + k, br, sizeof(struct branch));
				k++;
			}
			while (stack_pos > 0) {
				stack_pos--;
				nd = stack_node[stack_pos];
				i = stack_idx[stack_pos] + 1;

				if (i < nd->count_child_nodes) {
					// insert to stack
					stack_idx[stack_pos] = i;
					stack_node[stack_pos] = nd;
					stack_pos++;
					nd = (struct node*)nd->child_node[i];
					i = 0;
					break;
				}
			}
		}
		else {
			// insert to stack
			stack_idx[stack_pos] = i;
			stack_node[stack_pos] = nd;
			stack_pos++;
			nd = (struct node*)nd->child_node[i];
		}
	}
	struct branch *dev_br = NULL;
	er1 = hipMalloc((void**)&dev_br, sizeof(struct branch) * count_br);
	er1 = hipMemcpy(dev_br, tbr, sizeof(struct branch) * count_br, hipMemcpyHostToDevice);

	//return false;
	alignas(16) struct node *to_dev_nd[64];
	void **to_dev_child[64];
	struct node *dev_nd = NULL, *dev_nd_prev = NULL, *dev_ptr = NULL;
	// to_dev_nd[0] = (struct node*)aligned_alloc(16, sizeof(struct node));
	// memcpy(to_dev_nd[0], nd, sizeof(struct node));
	struct node* tnd = nd;
	for (unsigned j = 0; j <= pos; ++j)
		tnd = (struct node*)(tnd->child_node[0]);
	//unsigned j = 0;
	//void* tmp1 = NULL;
	unsigned count = tnd->count_child_nodes, prev_count = 1;
	for (int k1 = pos; k1 >= 0; --k1) {
		// data child node
		to_dev_nd[k1] = (struct node*)aligned_alloc(16, sizeof(struct node) * count1[k1]);
		memcpy(to_dev_nd[k1], tnd/*->child_node[0]*/, sizeof(struct node) * count1[k1]);
		// pointer to child_node on host
		for (indexer j = 0; j < count1[k1]; ++j) {
			//(to_dev_nd[k1])[j]->child_node = (void**)aligned_alloc(16, sizeof(void*) * MAX_NODES); // tnd->count_child_nodes);
			//(to_dev_child[k1])[j] = to_dev_nd[j]->child_node;
			dev_ptr = NULL;
			er1 = hipMalloc((void**)&dev_ptr, sizeof(void*) * MAX_NODES);
			(to_dev_nd[k1])[j].child_node = (void**)dev_ptr;
			for (indexer k2 = 0; k2 < MAX_NODES; ++k2) {
				if (k1 == pos) {
					// copy pointer of branches
					struct branch *ptr = &(dev_br[k2 + j * MAX_NODES]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct branch*), hipMemcpyHostToDevice);
				}
				else {
					// copy pointer of nodes
					struct node* ptr = &(dev_nd_prev[k2 + j * MAX_NODES]);
					er1 = hipMemcpy((void*)((to_dev_nd[k1])[j].child_node + k2), &ptr, sizeof(struct node*), hipMemcpyHostToDevice);
				}
			}
		}
		// pointers of child nodes
		er1 = hipMalloc((void**)&dev_nd, sizeof(struct node*) * count1[k1]); // tnd->count_child_nodes);
		hipMemcpy(dev_nd, to_dev_nd[k1], sizeof(struct node) * count1[k1], hipMemcpyHostToDevice);
		dev_nd_prev = dev_nd;
		continue;

		indexer k3 = 0;
		for (unsigned k2 = 0; k2 < count1[k1]; ++k2) {
			for (indexer j = 0; j < MAX_NODES; ++j) {
				((to_dev_nd[k1])[k2]).child_node[j] = &(dev_nd[k3]);
				k3++;
			}
		}


		break;
		// copy to device
		//hipMemcpy(dev_nd, to_dev_nd[j]->child_node[0], sizeof(struct node*) * count, hipMemcpyHostToDevice);
		// pointer to child_node on device
	/*	er1 = hipMalloc((void**)&dev_nd, sizeof(void*) * count);
		to_dev_nd[j]->child_node = (void**)dev_nd;
		// copy to device child_node
		//hipMemcpy(dev_nd, to_dev_nd[j]->child_node, sizeof(void*) * count, hipMemcpyHostToDevice);

		// prepare device to copy nodes
		//hipMalloc((void**)&dev_nd, sizeof(struct node) * prev_count);
		// copy to device nodes
		//hipMemcpy(dev_nd, tnd, sizeof(struct node) * prev_count, hipMemcpyHostToDevice);
		if (j == 1) {
			//to_dev_nd[0]->
		}

		prev_count = count;

		// free local memory
		//_aligned_free(tmp1); // to_dev_nd[j]->child_node);
		//_aligned_free(to_dev_nd[j]);

		j++;
		count = 0;
		if (!((struct node*)tnd->child_node[0])->is_last_node)
			for (unsigned k = 0; k < tnd->count_child_nodes; ++k) {
				count += ((struct node*)(tnd->child_node[k]))->count_child_nodes;
			}
		tnd = (struct node*)(tnd->child_node[0]);*/
	}

	// allocating memory for root
	//er1 = hipMalloc((void**)&m_dev_node, sizeof(struct node));
	// copy to device root of tree
	//hipMemcpy(m_dev_node, to_dev_nd[0], sizeof(struct node), hipMemcpyHostToDevice);

	return true;
}

extern "C"
bool destroy_cuda_device()
{
	hipError_t er1 = hipDeviceReset();
	return er1 == hipSuccess ? true : false;
}